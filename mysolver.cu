#include "hip/hip_runtime.h"
/* cuda network simulation 
   History :                    
    created: Shrisha
   Makefile included for build on CC=3.5
*/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "globalVars.h"
#include "aux.cu"
#include "cuda_histogram.h"

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}
void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

int main(int argc, char *argv[]) {
  double tStart = 0.0, tStop =  3000.0;
  double *spkTimes, *vm = NULL, host_theta = 0.0, theta_degrees; /* *vstart; 500 time steps */
  int *nSpks, *spkNeuronIds, nSteps, i, k, lastNStepsToStore;
  double *dev_vm = NULL, *dev_spkTimes, *dev_time = NULL, *host_time;
  int *dev_conVec = NULL, *dev_nSpks, *dev_spkNeuronIds;
  FILE *fp, *fpConMat, *fpSpkTimes, *fpElapsedTime;
  double *host_isynap, *synapticCurrent = NULL;
  /*  int *conVec;*/
  hiprandState *devStates, *devNormRandState, *poisRandState;;
  hipEvent_t start0, stop0;
  float elapsedTime;
  int *dev_sparseVec = NULL, *sparseConVec = NULL;
  int *dev_sparseVecFF = NULL, *sparseConVecFF = NULL;
  int idxVec[N_NEURONS], nPostNeurons[N_NEURONS], *dev_idxVec = NULL, *dev_nPostneuronsPtr = NULL;
  int *idxVecFF = NULL, *nPostNeuronsFF = NULL, *dev_idxVecFF = NULL, *dev_nPostneuronsPtrFF = NULL;
  int deviceId = 0;
  devPtr_t devPtrs;
  kernelParams_t kernelParams;
  int IF_SAVE = 1;
  hipStream_t stream1;
  char filetag[16];
  cudaCheck(hipStreamCreate(&stream1));
  printf("old tstop = %f\n", tStop);
  idxVecFF = (int *)malloc((unsigned long long)NFF * sizeof(int));
  nPostNeuronsFF = (int *)malloc((unsigned long long)NFF * sizeof(int));
  //  cudaCheck(hipMalloc((void **)&idxVecFF,  NFF * sizeof(*idxVecFF)));
  //  cudaCheck(hipMalloc((void **)&nPostNeuronsFF,  NFF * sizeof(*nPostNeuronsFF)));
  /*PARSE INPUTS*/
  if(argc > 1) {
    deviceId = atoi(argv[1]);
    if(argc > 2) {
      IF_SAVE = atoi(argv[2]);
    }
    if(argc > 3) {
      host_theta = atof(argv[3]);
    }
    if(argc > 4) {
      //      tStop = tStop + atof(argv[4]);
      strcpy(filetag, argv[4]);
    }
  }
  printf("\n Computing on GPU%d \n", deviceId);
  cudaCheck(hipSetDevice(deviceId));
  theta_degrees = host_theta;
  host_theta = PI * host_theta / (180.0); /* convert to radians */
  hipMemcpyToSymbol(HIP_SYMBOL(theta), &host_theta, sizeof(host_theta));
  /* ================= INITIALIZE ===============================================*/
  nSteps = (tStop - tStart) / DT;
  lastNStepsToStore = (int)floor(STORE_LAST_T_MILLISEC  / DT);
  //  nSteps = 800;
  printf("\n N  = %llu \n NE = %llu \n NI = %llu \n K  = %d \n tStop = %d milli seconds nSteps = %d\n\n", N_NEURONS, NE, NI, (int)K, (int)tStop, nSteps);
  
  printf(" theta = %2.3f \n contrast = %2.1f\n ksi = %f\n dt = %f \n tau = %f \n EXP_SUM = %.16f\n", host_theta, HOST_CONTRAST, ETA_E, DT, TAU_SYNAP, EXP_SUM);
  printf("alpha = %f, RHO = %f\n", ALPHA, RHO);
  
  // STORE MEAN G_FF
  double *dev_GFFmeanPtr = NULL, *host_GFFmean = NULL; 
  unsigned long long *devGFFCounterPtr = NULL, *hostGFFCounter = NULL;
  //  cudaCheck(hipMalloc((void **)&dev_GFFmean,  N_NEURONS * sizeof(*dev_GFFmean)));
  cudaCheck(hipGetSymbolAddress((void **)&dev_GFFmeanPtr, dev_GFFmean));
  cudaCheck(hipGetSymbolAddress((void **)&devGFFCounterPtr, devGFFCounter));
  cudaCheck(hipHostMalloc((void **)&host_GFFmean,  N_NEURONS * sizeof(*host_GFFmean)));
  cudaCheck(hipHostMalloc((void **)&hostGFFCounter, sizeof(*hostGFFCounter)));
  *hostGFFCounter = 0;


  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = 128;
  int BlocksPerGrid = (N_NEURONS + ThreadsPerBlock - 1) / ThreadsPerBlock;
  printf("Threads per block : %d, Blocks per grid : %d \n", ThreadsPerBlock, BlocksPerGrid);
  /*INITIALIZE RND GENERATORS FOR ibf & iff */
  setupBGCurGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  setupIFFRndGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  /*Generate frozen FF input approximat*/
  cudaCheck(hipMalloc((void **)&devStates,  N_NEURONS * sizeof(hiprandState)));

  unsigned long long tttt = 45687ULL;
  cudaCheck(hipMalloc((void **)&devNormRandState, N_NEURONS * sizeof(hiprandState)));
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, tttt);
  tttt = 12463ULL; //this has to be fixed so that the input structure is kept the same when running multiple simulations, or else each realization will give different tuning curvs 
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, tttt);
  AuxRffTotal<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, devStates);
  cudaCheck(hipFree(devNormRandState));
  /* SETUP POISSION RAND GENERATOR */
  cudaCheck(hipMalloc((void **)&poisRandState,  NFF * sizeof(hiprandState)));
  setup_pois_kernel<<<(NFF + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(poisRandState, time(NULL));
  AuxRffTotalWithOriMap<<<(NFF + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(); // assign POs in the ori map for the poiss generators 

  cudaCheck(hipGetSymbolAddress((void **)&dev_sparseVec, dev_sparseConVec));
  cudaCheck(hipGetSymbolAddress((void **)&dev_idxVec, dev_sparseIdx));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nPostneuronsPtr, dev_nPostNeurons));
  cudaCheck(hipHostMalloc((void **)&sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int)));

  cudaCheck(hipGetSymbolAddress((void **)&dev_sparseVecFF, dev_sparseConVecFF));
  cudaCheck(hipGetSymbolAddress((void **)&dev_idxVecFF, dev_sparseIdxFF));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nPostneuronsPtrFF, dev_nPostNeuronsFF));
  cudaCheck(hipHostMalloc((void **)&sparseConVecFF, N_NEURONS * (2ULL * (unsigned long long)(CFF * K) + 1) * sizeof(int)));

  FILE *fpSparseConVec, *fpIdxVec, *fpNpostNeurons;
  fpSparseConVec = fopen("sparseConVec.dat", "rb");
  fpIdxVec = fopen("idxVec.dat", "rb");
  fpNpostNeurons = fopen("nPostNeurons.dat", "rb");
  int dummy;
  dummy = fread(sparseConVec, sizeof(*sparseConVec), N_NEURONS * (2 * (int)K + 1), fpSparseConVec);
  if(dummy != N_NEURONS * (2 * (int)K + 1)) {
    printf("sparseConvec read error ? \n");
  }
  dummy = fread(idxVec, sizeof(*idxVec), N_NEURONS, fpIdxVec);
  dummy = fread(nPostNeurons, sizeof(*nPostNeurons), N_NEURONS, fpNpostNeurons);
  fclose(fpSparseConVec);
  fclose(fpIdxVec);
  fclose(fpNpostNeurons);
  /* READ FF SPARSE CONNECTION MAT */
  FILE *fpSparseConVecFF, *fpIdxVecFF, *fpNpostNeuronsFF;
  fpSparseConVecFF = fopen("sparseConVecFF.dat", "rb");
  fpIdxVecFF = fopen("idxVecFF.dat", "rb");
  fpNpostNeuronsFF = fopen("nPostNeuronsFF.dat", "rb");
  dummy = fread(sparseConVecFF, sizeof(*sparseConVecFF), N_NEURONS * (2ULL * (unsigned long long)(CFF * K) + 1), fpSparseConVecFF);
  if(dummy != N_NEURONS * (2 * (int)(CFF * K) + 1)) {
    printf("%d, sparseConvecFF read error ? \n", dummy);
  }
  dummy = fread(idxVecFF, sizeof(*idxVecFF), NFF, fpIdxVecFF);
  dummy = fread(nPostNeuronsFF, sizeof(*nPostNeuronsFF), NFF, fpNpostNeuronsFF);
  fclose(fpSparseConVecFF);
  fclose(fpIdxVecFF);
  fclose(fpNpostNeuronsFF);
  /*
    for(i = 0; i < N_NEURONS; ++i) {
      printf("neuron %d projects to : ", i);
      for(int j = 0; j < nPostNeurons[i]; ++j) {
	printf("%d ", sparseConVec[idxVec[i] + j]);
      }
      printf("\n");
    }
  */

  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVec, idxVec, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtr, nPostNeurons, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  printf("tst point, %p ----- %p \n", dev_sparseVec, dev_sparseVecFF);
  cudaCheck(hipMemcpy(dev_sparseVecFF, sparseConVecFF, N_NEURONS * (2ULL * (unsigned long long)(CFF * K) + 1)* sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVecFF, idxVecFF, NFF * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtrFF, nPostNeuronsFF, NFF * sizeof(int), hipMemcpyHostToDevice));
 /* ================= ALLOCATE PAGELOCKED MEMORY ON HOST =========================*/
  cudaCheck(hipHostMalloc((void **)&spkTimes, MAX_SPKS  * sizeof(*spkTimes)));
  cudaCheck(hipHostMalloc((void **)&host_isynap, N_I_SAVE_CUR * sizeof(*host_isynap)));
  cudaCheck(hipHostMalloc((void **)&vm,  lastNStepsToStore * N_NEURONS_TO_STORE * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&host_time,  lastNStepsToStore * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&nSpks, sizeof(*nSpks)));
  cudaCheck(hipHostMalloc((void **)&spkNeuronIds, MAX_SPKS * sizeof(*spkNeuronIds)));
  /* ================= ALLOCATE GLOBAL MEMORY ON DEVICE ===========================*/
  /*cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));*/
  cudaCheck(hipMalloc((void **)&dev_vm, lastNStepsToStore * N_NEURONS_TO_STORE * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_time, lastNStepsToStore * sizeof(double)));
  cudaCheck(hipMalloc((void **)&synapticCurrent, N_I_SAVE_CUR * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_spkTimes, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMalloc((void **)&dev_nSpks, sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_spkNeuronIds, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  cudaCheck(hipMemset(dev_spkTimes, 0, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMemset(dev_spkNeuronIds, 0.0f, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  printf(" GPU memory allocation successful ! \n ");
  devPtrs.dev_conVec = dev_conVec;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_vm = dev_vm;
  devPtrs.dev_nSpks = dev_nSpks;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_spkTimes = dev_spkTimes;
  devPtrs.synapticCurrent = synapticCurrent;
  devPtrs.devStates = devStates;
  /*  devPtrs.dev_sparseConVec = dev_sparseVec;
  devPtrs.dev_nPostNeurons = dev_nPostneuronsPtr;
  devPtrs.dev_sparseIdx = dev_idxVec;*/
  devPtrs.dev_time = dev_time;
  *nSpks = 0;
  cudaCheck(hipMemcpy(dev_nSpks, nSpks, sizeof(int), hipMemcpyHostToDevice));




  /*  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);*/
  /*===================== GENERATE CONNECTION MATRIX ====================================*/
  /*cudaCheck(hipMemset(dev_conVec, 0, N_NEURONS * N_NEURONS * sizeof(int)));
  printf("\n launching rand generator setup kernel\n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  printf("\n launching connection matrix geneting kernel with seed %ld ...", time(NULL));
  fflush(stdout);
  kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVec);
  printf(" Done! \n");
  cudaCheck(hipMemcpy(conVec, dev_conVec, N_NEURONS * N_NEURONS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipFree(dev_conVec));
  GenSparseMat(conVec, N_NEURONS, N_NEURONS, sparseConVec, idxVec, nPostNeurons);
  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVec, idxVec, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtr, nPostNeurons, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));*/
  /* ==================== INTEGRATE ODEs ON GPU ==========================================*/
    /* invoke device on this block/thread grid */
  kernelParams.nSteps = nSteps;
  kernelParams.tStop = tStop;
  kernelParams.tStart = tStart;
  printf("\n launching Simulation kernel ...");
  fflush(stdout);
  
  
  
  int *dev_IF_SPK_Ptr = NULL, *dev_prevStepSpkIdxPtr = NULL, *host_IF_SPK = NULL, *host_prevStepSpkIdx = NULL,  *dev_nEPtr = NULL, *dev_nIPtr = NULL, *dev_IF_SPK_POISSION_Ptr = NULL;
  int *host_FF_IF_SPK, nFFSpksInPrevStep;
  int nSpksInPrevStep;
  cudaCheck(hipHostMalloc((void **)&host_IF_SPK, N_NEURONS * sizeof(int)));
  cudaCheck(hipHostMalloc((void **)&host_prevStepSpkIdx, N_NEURONS * sizeof(int)));
  cudaCheck(hipGetSymbolAddress((void **)&dev_IF_SPK_Ptr, dev_IF_SPK));
  cudaCheck(hipGetSymbolAddress((void **)&dev_prevStepSpkIdxPtr, dev_prevStepSpkIdx));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nEPtr, dev_ESpkCountMat));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nIPtr, dev_ISpkCountMat));
  cudaCheck(hipGetSymbolAddress((void **)&dev_IF_SPK_POISSION_Ptr, IF_SPIKE_POISSION_SPK));
  cudaCheck(hipHostMalloc((void **)&host_FF_IF_SPK, NFF * sizeof(int)));
  printf("\n %p \n", host_FF_IF_SPK);
  cudaCheck(hipMemcpy(host_FF_IF_SPK, dev_IF_SPK_POISSION_Ptr, NFF * sizeof(int), hipMemcpyDeviceToHost));
  for(i = 0; i < N_NEURONS; ++i) {
    host_IF_SPK[i] = 0;
    if(i < NFF) {
      host_FF_IF_SPK[i] = 0;
    }
  }


  /* TIME LOOP */
  size_t sizeOfInt = sizeof(int);
  size_t sizeOfDbl = sizeof(double);
  /* SETUP TIMER EVENTS ON DEVICE */
  hipEventCreate(&stop0); hipEventCreate(&start0);
  hipEventRecord(start0, 0);
  unsigned int spksE = 0, spksI = 0, spksFF = 0;
  FILE *fpIFR = fopen("instant_fr.csv", "w");
  int *histVec = NULL, *dev_histVec = NULL, *histVecFF = NULL, *dev_histVecFF = NULL; /* for storing the post-synaptic neurons to be updated */
  int histVecIndx = 0, histVecIndxFF = 0;
  unsigned int histVecLength = N_NEURONS * (int)K;
  unsigned int histVecLengthFF = N_NEURONS * (int)(CFF * K);
  if((unsigned long long)K >= NE | (unsigned long long)K >= NI) {
    histVecLength = (unsigned int)(N_NEURONS * N_NEURONS);
  }
  cudaCheck(hipHostMalloc((void **)&histVec, histVecLength * sizeof(*histVec)));
  cudaCheck(hipMalloc((void **)&dev_histVec, histVecLength * sizeof(*dev_histVec)));

  cudaCheck(hipHostMalloc((void **)&histVecFF, histVecLengthFF * sizeof(*histVecFF)));
  cudaCheck(hipMalloc((void **)&dev_histVecFF, histVecLengthFF * sizeof(*dev_histVecFF)));

  test_xform xform; // defined in cuda_histogram.h
  test_sumfun sum;  // defined in cuda_histogram.h
  int *dev_histCountE = NULL, *histCountE = NULL, *dev_histCountI = NULL, *histCountI = NULL;
  int *dev_histCountFF = NULL, *histCountFF = NULL;
  cudaCheck(hipMalloc((void **)&dev_histCountE, sizeof(int) * N_NEURONS));
  cudaCheck(hipHostMalloc((void **)&histCountE, sizeof(int) * N_NEURONS));
  cudaCheck(hipMalloc((void **)&dev_histCountI, sizeof(int) * N_NEURONS));
  cudaCheck(hipHostMalloc((void **)&histCountI, sizeof(int) * N_NEURONS));

  cudaCheck(hipMalloc((void **)&dev_histCountFF, sizeof(int) * N_NEURONS));
  cudaCheck(hipHostMalloc((void **)&histCountFF, sizeof(int) * N_NEURONS));

  int tmp;
  char fileSuffix[128], filename[128];
  strcpy(filename, "currents");
  sprintf(fileSuffix, "_%1.1f_%1.f", ALPHA, TAU_SYNAP);
  strcat(filename, fileSuffix);
  FILE *fpCur = NULL;
  fpCur = fopen(strcat(filename, ".csv"), "w");
  /*printf("\n\n\n\n %d\n\n\n\n", sparseConVec[835584ULL]);*/


  for(k = 0; k < nSteps; ++k) { 
    /*    cudaCheck(hipMemsetAsync(dev_nEPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP * sizeOfInt, stream1));
	  cudaCheck(hipMemsetAsync(dev_nIPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP * sizeOfInt, stream1));*/
    /*    nSpksInPrevStep = 0;*/
    devPtrs.k = k;
    nSpksInPrevStep = 0;
    histVecIndx = 0;
    histVecIndxFF = 0;
    for(i = 0; i < N_NEURONS; ++i) {
      histCountI[i] = 0;
      histCountE[i] = 0;
      histCountFF[i] = 0;
    }

    GenPoissionSpikeInFFLayer<<<(NFF + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(poisRandState); // GENERATE SPIKES IN LAYER 4 
    rkdumbPretty<<<BlocksPerGrid, ThreadsPerBlock>>> (kernelParams, devPtrs);
    cudaCheckLastError("rk");
    if(k > 0) {
      /*      cudaCheck(hipMemcpy(host_IF_SPK, dev_IF_SPK_Ptr, N_NEURONS * sizeOfInt, hipMemcpyDeviceToHost));*/
      cudaCheck(hipMemcpyAsync(host_IF_SPK, dev_IF_SPK_Ptr, N_NEURONS * sizeOfInt, hipMemcpyDeviceToHost, stream1));
      cudaCheck(hipMemcpyAsync(host_isynap, synapticCurrent, N_I_SAVE_CUR * sizeOfDbl, hipMemcpyDeviceToHost, stream1));
      cudaCheck(hipMemcpyAsync(host_FF_IF_SPK, dev_IF_SPK_POISSION_Ptr, NFF * sizeOfInt, hipMemcpyDeviceToHost, stream1));
    }
    cudaCheck(hipStreamSynchronize(stream1));
    /*instantaneous firing rate, rect non-overlapping window */
    for(i = 0; i < N_NEURONS; ++i) {
      if(host_IF_SPK[i]) {
        if(i < NE) {
          spksE += 1;
        }
        else{
          spksI += 1;
        }
       
	/*	    host_prevStepSpkIdx[i] = nSpksInPrevStep;
		    nSpksInPrevStep += 1;*/
      }

      if(i < NFF) {
        if(host_FF_IF_SPK[i]) {
            spksFF += 1;
        }
      }
    }
    if(!(k%(int)(50.0/DT))) {
      fprintf(fpIFR, "%f %f \n", ((double)spksE) / (0.05 * (double)NE), ((double)spksI) / (0.05 * (double)NI));fflush(fpIFR);
      fprintf(stdout, "%f %f ", ((double)spksE) / (0.05 * (double)NE), ((double)spksI) / (0.05 * (double)NI));
      spksE = 0; 
      spksI = 0;
      fprintf(stdout, "%f \n", ((double)spksFF) / (0.05 * (double)NFF));
      spksFF= 0;
    }
    /*-----------------------------------------------------------------------*/
    expDecay<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_histCountE, dev_histCountI);
    expDecayGFF<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_histCountFF);
    cudaCheckLastError("exp");
    for(i = 0; i < NE; ++i) {
      if(host_IF_SPK[i]){
      nSpksInPrevStep += 1;
        for(int jj = 0; jj < nPostNeurons[i]; ++jj) {
          tmp = sparseConVec[idxVec[i] + jj];
          histVec[histVecIndx++] = tmp;
            /*          histVec[histVecIndx++] = sparseConVec[idxVec[i] + jj];*/
        }
      }
    }
    if(nSpksInPrevStep) {
      cudaCheck(hipMemcpy(dev_histVec, histVec, histVecIndx * sizeof(int), hipMemcpyHostToDevice));
      callHistogramKernel<histogram_atomic_inc, 1>(dev_histVec, xform, sum, 0, histVecIndx, 0, &histCountE[0], (int)N_NEURONS);
      /*      cudaCheck(hipMemcpy(dev_histCountE, histCountE, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));*/
      cudaCheck(hipMemcpyAsync(dev_histCountE, histCountE, N_NEURONS * sizeof(int), hipMemcpyHostToDevice, stream1));
    }
    
    histVecIndx = 0;
    nSpksInPrevStep = 0; 
    for(i = NE; i < N_NEURONS; ++i) {
      if(host_IF_SPK[i]){
        nSpksInPrevStep += 1;
        for(int jj = 0; jj < nPostNeurons[i]; ++jj) {
          histVec[histVecIndx++] = sparseConVec[idxVec[i] + jj];
        }
      }
    }
    
    if(nSpksInPrevStep) {
      cudaCheck(hipMemcpy(dev_histVec, histVec, histVecIndx * sizeof(int), hipMemcpyHostToDevice));
      callHistogramKernel<histogram_atomic_inc, 1>(dev_histVec, xform, sum, 0, histVecIndx, 0, &histCountI[0], (int)N_NEURONS);
      cudaCheckLastError("HIST");
      cudaCheck(hipMemcpy(dev_histCountI, histCountI, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
      /*      cudaCheck(hipMemcpyAsync(dev_histCountI, histCountI, N_NEURONS * sizeof(int), hipMemcpyHostToDevice, stream1));*/
    }

    histVecIndxFF = 0;
    nFFSpksInPrevStep = 0;
    for(i = 0; i < NFF; ++i) {
      if(host_FF_IF_SPK[i]){
        nFFSpksInPrevStep += 1;
        for(int jj = 0; jj < nPostNeuronsFF[i]; ++jj) {
          histVec[histVecIndxFF++] = sparseConVecFF[idxVecFF[i] + jj];
        }
      }
    }
    if(nFFSpksInPrevStep) {
      cudaCheck(hipMemcpy(dev_histVecFF, histVec, histVecIndxFF * sizeof(int), hipMemcpyHostToDevice));
      callHistogramKernel<histogram_atomic_inc, 1>(dev_histVecFF, xform, sum, 0, histVecIndxFF, 0, &histCountFF[0], (int)N_NEURONS);
      cudaCheckLastError("HIST_FF");
      cudaCheck(hipMemcpy(dev_histCountFF, histCountFF, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
    }

    // int tmpCnt = 0;
    // for(i = 0; i < N_NEURONS; ++i) {
    //   if(histCountFF[i]) {
    //     tmpCnt += histCountFF[i];
    //   }
    // }
    //    printf("histcount = %d\n", tmpCnt);
    /*    expDecay<<<BlocksPerGrid, ThreadsPerBlock>>>();*/

    /*computeConductance<<<BlocksPerGrid, ThreadsPerBlock>>>();*/
    cudaCheck(hipStreamSynchronize(stream1));

    /* SAVE CURRENT VALUES TO DISK  */
    // for(int jj = 0; jj < N_I_SAVE_CUR; ++jj) {
    //   fprintf(fpCur, "%f ", host_isynap[jj]);
    // }
    // fprintf(fpCur, "\n";)
    computeConductanceHist<<<(N_NEURONS + 512 - 1) / 512, 512>>>(dev_histCountE, dev_histCountI);
    computeConductanceHistFF<<<(N_NEURONS + 512 - 1) / 512, 512>>>(dev_histCountFF);
    cudaCheckLastError("g");
    computeIsynap<<<BlocksPerGrid, ThreadsPerBlock>>>(k*DT);
    cudaCheckLastError("isyp");
  }
  fclose(fpCur);
  cudaCheck(hipHostFree(histVec));
  cudaCheck(hipFree(dev_histVec));
  cudaCheck(hipFree(dev_histCountE));
  cudaCheck(hipFree(dev_histCountI));
  cudaCheck(hipHostFree(histCountE));  
  cudaCheck(hipHostFree(histCountI));
  fclose(fpIFR);
  cudaCheck(hipStreamDestroy(stream1));
  cudaCheckLastError("rkdumb kernel failed");
  hipEventRecord(stop0, 0);
  hipEventSynchronize(stop0);
  printf(" Done ! \n");
  hipEventElapsedTime(&elapsedTime, start0, stop0);
  printf("\n elapsed time = %fms \n", elapsedTime);
  cudaCheck(hipEventDestroy(start0));
  cudaCheck(hipEventDestroy(stop0));
  /*==================== COPY RESULTS TO HOST =================================================*/
  cudaCheck(hipMemcpy(nSpks, dev_nSpks, sizeof(int), hipMemcpyDeviceToHost));
  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);
  cudaCheck(hipMemcpy(spkTimes, dev_spkTimes, MAX_SPKS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(spkNeuronIds, dev_spkNeuronIds, MAX_SPKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(host_time, dev_time, lastNStepsToStore * sizeof(double), hipMemcpyDeviceToHost));
  /*  cudaCheck(hipMemcpy(host_isynap, synapticCurrent, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));*/
  /*  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));*/
  double curE[N_CURRENT_STEPS_TO_STORE], curI[N_CURRENT_STEPS_TO_STORE], ibgCur[N_CURRENT_STEPS_TO_STORE], *dev_curE, *dev_curI, *dev_ibg, curIff[N_CURRENT_STEPS_TO_STORE], *dev_curiff;
  cudaCheck(hipGetSymbolAddress((void **)&dev_curE, glbCurE));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curI, glbCurI));
  cudaCheck(hipGetSymbolAddress((void **)&dev_ibg, dev_bgCur));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curiff, dev_iff));
  cudaCheck(hipMemcpy(curE, dev_curE, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curI, dev_curI, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(ibgCur, dev_ibg, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curIff, dev_curiff, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  printf("\n nSpks = %d\n", *nSpks);
  printf(" MAX SPKS stored on GPU = %d \n", MAX_SPKS); 
  printf("\n Simulation completed ! \n");
  fpElapsedTime = fopen("elapsedTime.csv", "a+");
  fprintf(fpElapsedTime, "%llu %f %d\n", N_NEURONS, elapsedTime, *nSpks);
  fclose(fpElapsedTime);
  /* ================= SAVE TO DISK =============================================================*/
  printf(" saving results to disk ... "); 
  fflush(stdout);
  //  char fileSuffix[128], filename[128];

  strcpy(filename, "spkTimes");
  sprintf(fileSuffix, "_xi%1.1f_theta%d_%.2f_%1.1f_%d_tr%s", ETA_E, (int)theta_degrees, ALPHA, TAU_SYNAP, (int)(tStop), filetag);
  strcat(filename, fileSuffix);
  fpSpkTimes = fopen(strcat(filename, ".csv"),"w");
  /*  fpSpkTimes = fopen("spkTimes.csv", "w");*/
  int totalNSpks = *nSpks;
  printf(" saving spikes ...");
  fflush(stdout);
  if(*nSpks > MAX_SPKS) {
    totalNSpks = MAX_SPKS;
    printf("\n ***** WARNING MAX_SPKS EXCEEDED limit of %d *****\n", MAX_SPKS);
  }
  for(i = 1; i <= totalNSpks; ++i) {
    fprintf(fpSpkTimes, "%f;%f\n", spkTimes[i], (double)spkNeuronIds[i]);
  }
  fclose(fpSpkTimes);
  printf("done\n");

  cudaCheck(hipMemcpy(host_GFFmean, dev_GFFmeanPtr, N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(hostGFFCounter, devGFFCounterPtr, sizeof(unsigned long long), hipMemcpyDeviceToHost));
  /*  SAVE CONDUCTANCES */
  if(nSteps > 20000) {
    strcpy(filename, "gffmean");
    sprintf(fileSuffix, "_R0%1.1f_theta%d_%.2f_%1.1f_%d_tr%s", R0, (int)theta_degrees, ALPHA, TAU_SYNAP, (int)(tStop), filetag);
    strcat(filename, fileSuffix);
    FILE* fpGFFmean;
    fpGFFmean = fopen(strcat(filename, ".csv"),"w");
    double denom = 0.0;
    denom = (double)hostGFFCounter[0];
    for(k = 0; k < N_NEURONS; ++k) {
      fprintf(fpGFFmean, "%f\n", (double)host_GFFmean[k] / denom);
    }
    fclose(fpGFFmean);
  }

  printf("saving vm to disk ....");
  fflush(stdout);
  if(IF_SAVE) {
    //    char fileSuffix[128], filename[128];
    strcpy(filename, "vm");
    sprintf(fileSuffix, "_xi%1.1f_theta%d_%.2f_%1.1f_%d_tr%s", ETA_E, (int)theta_degrees, ALPHA, TAU_SYNAP, (int)(tStop), filetag);
    //sprintf(fileSuffix, "_%1.1f_%1.1f", ALPHA, TAU_SYNAP);
    strcat(filename, fileSuffix);
    fp = fopen(strcat(filename, ".csv"),"w");
    //    fp = fopen("vm.csv", "w");
    for(i = 0; i < lastNStepsToStore; ++i) {
      fprintf(fp, "%f ", host_time[i]);
      for(k = 0; k < N_NEURONS_TO_STORE; ++k) {
	/*	fprintf(fp, "%f %f ", vm[k + i *  N_NEURONS], host_isynap[k + i * N_NEURONS]);*/
        fprintf(fp, "%f ", vm[k + i *  N_NEURONS_TO_STORE]);
      }
      fprintf(fp, "\n");
    }
    printf("\n%d %d\n", i, k);
    fclose(fp);
    FILE* fpCur = fopen("currents.csv", "w");
    for(i = 0; i < N_CURRENT_STEPS_TO_STORE; ++i) {
      fprintf(fpCur, "%f;%f;%f;%f\n", curE[i], curI[i], ibgCur[i], curIff[i]);
    //      fprintf(fpCur, "%f\n", curIff[i]);
    }
    fclose(fpCur);
    fpConMat = fopen("conMat.csv", "w");
    fpConMat = fopen("conVec.csv", "w");

    /*    for(i = 0; i < N_NEURONS; ++i) {
      for(k = 0; k < N_NEURONS; ++k) {
	fprintf(fpConMat, "%d", conVec[i *  N_NEURONS + k]);
      }
            fprintf(fpConMat, "\n");

      }*/
    fclose(fpConMat);
  }
  printf("done\n");
  /*================== CLEANUP ===================================================================*/
  cudaCheck(hipHostFree(vm));
  cudaCheck(hipHostFree(host_time));
  /*  cudaCheck(hipHostFree(host_isynap));*/
  cudaCheck(hipHostFree(spkTimes));
  cudaCheck(hipHostFree(spkNeuronIds));
  cudaCheck(hipHostFree(nSpks));
  cudaCheck(hipFree(dev_vm));
  cudaCheck(hipFree(dev_time));
  /*  cudaCheck(hipFree(synapticCurrent));*/
  cudaCheck(hipFree(dev_spkNeuronIds));
  cudaCheck(hipFree(dev_spkTimes));
  cudaCheck(hipFree(dev_nSpks));
  cudaCheck(hipFree(devStates));
  /*  cudaCheck(hipFree(dev_sparseVec));
  cudaCheck(hipFree(dev_idxVec));
  cudaCheck(hipFree(dev_nPostneuronsPtr));*/
  cudaCheck(hipHostFree(host_IF_SPK));
  cudaCheck(hipHostFree(host_prevStepSpkIdx));

  cudaCheck(hipHostFree(host_GFFmean));
  cudaCheck(hipHostFree(hostGFFCounter));

  free(idxVecFF);
  free(nPostNeuronsFF);
  // cudaCheck(hipHostFree(idxVecFF));
  // cudaCheck(hipHostFree(nPostNeuronsFF));
  /*  hipDeviceReset()*/
  return EXIT_SUCCESS;
}

