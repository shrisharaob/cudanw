#include "hip/hip_runtime.h"
/* cuda network simulation 
   History :                    
    created: Shrisha
   Makefile included for build on CC=3.5
*/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "globalVars.h"
#include "aux.cu"
#include "cuda_histogram.h"

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}
void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

int main(int argc, char *argv[]) {
  double tStart = 0.0, tStop = 100000.0;
  double *spkTimes, *vm = NULL, host_theta = 0.0, theta_degrees; /* *vstart; 500 time steps */
  int *nSpks, *spkNeuronIds, nSteps, i, k, lastNStepsToStore;
  double *dev_vm = NULL, *dev_spkTimes, *dev_time = NULL, *host_time;
  int *dev_conVec = NULL, *dev_nSpks, *dev_spkNeuronIds;
  FILE *fp, *fpConMat, *fpSpkTimes, *fpElapsedTime;
  double *host_isynap, *synapticCurrent = NULL;
  /*  int *conVec;*/
  hiprandState *devStates, *devNormRandState;
  hipEvent_t start0, stop0;
  float elapsedTime;
  int *dev_sparseVec = NULL, *sparseConVec = NULL;
  int idxVec[N_NEURONS], nPostNeurons[N_NEURONS], *dev_idxVec = NULL, *dev_nPostneuronsPtr = NULL;
  int deviceId = 0;
  devPtr_t devPtrs;
  kernelParams_t kernelParams;
  int IF_SAVE = 1;
  hipStream_t stream1;
  char filetag[16];
  double *firingrate;

  printf("\n \n bg I  = %f \n", K*K_REC_I_PREFACTOR*G_IB*RB_I);
  firingrate = (double *) malloc(sizeof(double) * N_NEURONS);
  cudaCheck(hipStreamCreate(&stream1));
  //  printf("old tstop = %f\n", tStop);
  /*PARSE INPUTS*/
  if(argc > 1) {
    deviceId = atoi(argv[1]);
    if(argc > 2) {
      IF_SAVE = atoi(argv[2]);
    }
    if(argc > 3) {
      host_theta = atof(argv[3]);
    }
    if(argc > 4) {
      //      tStop = tStop + atof(argv[4]);
      strcpy(filetag, argv[4]);
    }
  }
  printf("\n Computing on GPU%d \n", deviceId);
  cudaCheck(hipSetDevice(deviceId));
  theta_degrees = host_theta;
  host_theta = PI * host_theta / (180.0); /* convert to radians */
  /*======SIMULATING EYE TITLE ========== */
  // srand(time(NULL));
  // double tmprnd = ((double) rand() / (RAND_MAX + 1.0)) * (5.0) - (2.5); // simulatinge eye tilt
  // host_theta += (tmprnd * PI / 180.0);
  /* ================= INITIALIZE ===============================================*/
  hipMemcpyToSymbol(HIP_SYMBOL(theta), &host_theta, sizeof(host_theta));
  nSteps = (tStop - tStart) / DT;
  lastNStepsToStore = (int)floor(STORE_LAST_T_MILLISEC  / DT);
  //  nSteps = 800;
  printf("\n N  = %llu \n NE = %llu \n NI = %llu \n KE  = %d, KI = %d \n tStop = %d milli seconds nSteps = %d\n\n", N_NEURONS, NE, NI, (int)K, (int)(K * K_REC_I_PREFACTOR), (int)tStop, nSteps);
  printf("KFF_E = %d, KFF_I = %d", (int)(CFFE * K), (int)(CFFI * K));
  
  printf(" theta = %2.3f \n contrast = %2.1f\n ksi = %f\n dt = %f \n tau E = %f , tau I = %f \n EXP_SUM_E = %.16f, EXP_SUM_I = %.16f\n Conductance glb prefactor = %f", host_theta * 180.0 / PI, HOST_CONTRAST, ETA_E, DT, TAU_SYNAP_E, TAU_SYNAP_I, EXP_SUM_E, EXP_SUM_I, CONDUCTANCE_GLOBAL_PREFACTOR);
  printf("\n alpha = %f, RHO = %f\n", ALPHA, RHO);
  
  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = 128;
  int BlocksPerGrid = (N_NEURONS + ThreadsPerBlock - 1) / ThreadsPerBlock;
  printf(" Threads per block : %d, Blocks per grid : %d \n", ThreadsPerBlock, BlocksPerGrid);
  /*INITIALIZE RND GENERATORS FOR ibf & iff */
  setupBGCurGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  setupIFFRndGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  /*Generate frozen FF input approximat*/
  cudaCheck(hipMalloc((void **)&devStates,  N_NEURONS * sizeof(hiprandState)));
  unsigned long long tttt = 45687ULL;
  cudaCheck(hipMalloc((void **)&devNormRandState, N_NEURONS * sizeof(hiprandState)));
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, tttt);
  tttt = 12463ULL; //this has to be fixed so that the input structure is kept the same when running multiple simulations, or else each realization will give different tuning curvs 
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, tttt);
  AuxRffTotal<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, devStates);
  cudaCheck(hipFree(devNormRandState));
  /* gENERATE CONNECTION MATRIX */
  /*  cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));*/
  /*  cudaCheck(hipHostMalloc((void **)&conVec, N_NEURONS * N_NEURONS * sizeof(int)));  */
  /*  cudaCheck(hipMemset(dev_conVec, 0, N_NEURONS * N_NEURONS * sizeof(int)));
  printf("\n launching rand generator setup kernel\n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  printf("\n launching connection matrix geneting kernel with seed %ld ...", time(NULL));
  fflush(stdout);
  kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVec);
  printf(" Done! \n");
  cudaCheck(hipMemcpy(conVec, dev_conVec, N_NEURONS * N_NEURONS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipFree(dev_conVec));*/
  /*  printf("reading convec.dat..."); fflush(stdout);
  FILE *fpConVecFile = fopen("conVec.dat", "rb");
  fread(conVec, sizeof(*conVec), N_NEURONS * N_NEURONS, fpConVecFile);
  fclose(fpConVecFile);
  printf("done ...\n");*/
  /* SPARSIFY */
  /*  conVec[0] = 0; conVec[1] = 0; conVec[2] = 1;conVec[3] = 0;*/
  /*conVec[4] = 0;conVec[5] = 1;conVec[6] = 1;conVec[7] = 1;
  conVec[8] = 1;*/ /*conVec[9] = 0;*/
  /*conVec[10] = 0;conVec[11] = 1;
    conVec[12]= 0;conVec[13] = 0;conVec[14] = 0;conVec[15] = 0;*/

  //  cudaCheck(hipGetSymbolAddress((void **)&dev_sparseVec, dev_sparseConVec));
  cudaCheck(hipGetSymbolAddress((void **)&dev_idxVec, dev_sparseIdx));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nPostneuronsPtr, dev_nPostNeurons));
  /*  cudaCheck(hipMalloc((void **)&dev_sparseVec, N_NEURONS * ((int)2 * K + 1)* sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_idxVec, N_NEURONS * sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_nPostneuronsPtr, N_NEURONS * sizeof(int)));*/
  /*  GenSparseMat(conVec, N_NEURONS, N_NEURONS, sparseConVec, idxVec, nPostNeurons);*/
  FILE *fpSparseConVec, *fpIdxVec, *fpNpostNeurons;
  fpSparseConVec = fopen("sparseConVec.dat", "rb");
  fpIdxVec = fopen("idxVec.dat", "rb");
  fpNpostNeurons = fopen("nPostNeurons.dat", "rb");
  int dummy;
  dummy = fread(nPostNeurons, sizeof(*nPostNeurons), N_NEURONS, fpNpostNeurons);
  fclose(fpNpostNeurons);
  unsigned long int nConnections = 0;
  for(i = 0; i < N_NEURONS; ++i) {
    nConnections += nPostNeurons[i];
  }
  cudaCheck(hipHostMalloc((void **)&sparseConVec, nConnections * sizeof(int)));
  //  cudaCheck(hipHostMalloc((void **)&sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_sparseVec,  nConnections * sizeof(int)));  
  //  dummy = fread(sparseConVec, sizeof(*sparseConVec), N_NEURONS * (2 * (int)K + 1), fpSparseConVec);
  dummy = fread(sparseConVec, sizeof(*sparseConVec), nConnections, fpSparseConVec);
  devPtrs.dev_sparseConVec = dev_sparseVec;
  if(dummy != nConnections) {
    printf("sparseConvec read error ? \n");
  }
  dummy = fread(idxVec, sizeof(*idxVec), N_NEURONS, fpIdxVec);
  
  fclose(fpSparseConVec);
  fclose(fpIdxVec);
  /*
    for(i = 0; i < N_NEURONS; ++i) {
      printf("neuron %d projects to : ", i);
      for(int j = 0; j < nPostNeurons[i]; ++j) {
	printf("%d ", sparseConVec[idxVec[i] + j]);
      }
      printf("\n");
    }
  */

  //  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, nConnections * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVec, idxVec, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtr, nPostNeurons, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
 /* ================= ALLOCATE PAGELOCKED MEMORY ON HOST =========================*/
  cudaCheck(hipHostMalloc((void **)&spkTimes, MAX_SPKS  * sizeof(*spkTimes)));
  cudaCheck(hipHostMalloc((void **)&host_isynap, N_I_SAVE_CUR * sizeof(*host_isynap)));
  cudaCheck(hipHostMalloc((void **)&vm,  lastNStepsToStore * N_NEURONS_TO_STORE * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&host_time,  lastNStepsToStore * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&nSpks, sizeof(*nSpks)));
  cudaCheck(hipHostMalloc((void **)&spkNeuronIds, MAX_SPKS * sizeof(*spkNeuronIds)));
  /* ================= ALLOCATE GLOBAL MEMORY ON DEVICE ===========================*/
  /*cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));*/
  cudaCheck(hipMalloc((void **)&dev_vm, lastNStepsToStore * N_NEURONS_TO_STORE * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_time, lastNStepsToStore * sizeof(double)));
  cudaCheck(hipMalloc((void **)&synapticCurrent, N_I_SAVE_CUR * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_spkTimes, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMalloc((void **)&dev_nSpks, sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_spkNeuronIds, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  cudaCheck(hipMemset(dev_spkTimes, 0, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMemset(dev_spkNeuronIds, 0.0f, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  printf(" GPU memory allocation successful ! \n ");
  devPtrs.dev_conVec = dev_conVec;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_vm = dev_vm;
  devPtrs.dev_nSpks = dev_nSpks;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_spkTimes = dev_spkTimes;
  devPtrs.synapticCurrent = synapticCurrent;
  devPtrs.devStates = devStates;
  /*  devPtrs.dev_sparseConVec = dev_sparseVec;
  devPtrs.dev_nPostNeurons = dev_nPostneuronsPtr;
  devPtrs.dev_sparseIdx = dev_idxVec;*/
  devPtrs.dev_time = dev_time;
  *nSpks = 0;
  cudaCheck(hipMemcpy(dev_nSpks, nSpks, sizeof(int), hipMemcpyHostToDevice));

  /*  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);*/
  /*===================== GENERATE CONNECTION MATRIX ====================================*/
  /*cudaCheck(hipMemset(dev_conVec, 0, N_NEURONS * N_NEURONS * sizeof(int)));
  printf("\n launching rand generator setup kernel\n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  printf("\n launching connection matrix geneting kernel with seed %ld ...", time(NULL));
  fflush(stdout);
  kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVec);
  printf(" Done! \n");
  cudaCheck(hipMemcpy(conVec, dev_conVec, N_NEURONS * N_NEURONS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipFree(dev_conVec));
  GenSparseMat(conVec, N_NEURONS, N_NEURONS, sparseConVec, idxVec, nPostNeurons);
  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVec, idxVec, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtr, nPostNeurons, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));*/
  /* ==================== INTEGRATE ODEs ON GPU ==========================================*/
    /* invoke device on this block/thread grid */
  kernelParams.nSteps = nSteps;
  kernelParams.tStop = tStop;
  kernelParams.tStart = tStart;
  printf("\n launching Simulation kernel ... \n");
  fflush(stdout);
  
  
  
  int *dev_IF_SPK_Ptr = NULL, *dev_prevStepSpkIdxPtr = NULL, *host_IF_SPK = NULL, *host_prevStepSpkIdx = NULL,  *dev_nEPtr = NULL, *dev_nIPtr = NULL;
  int nSpksInPrevStep;
  cudaCheck(hipHostMalloc((void **)&host_IF_SPK, N_NEURONS * sizeof(int)));
  cudaCheck(hipHostMalloc((void **)&host_prevStepSpkIdx, N_NEURONS * sizeof(int)));
  cudaCheck(hipGetSymbolAddress((void **)&dev_IF_SPK_Ptr, dev_IF_SPK));
  cudaCheck(hipGetSymbolAddress((void **)&dev_prevStepSpkIdxPtr, dev_prevStepSpkIdx));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nEPtr, dev_ESpkCountMat));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nIPtr, dev_ISpkCountMat));
  for(i = 0; i < N_NEURONS; ++i) {
    host_IF_SPK[i] = 0;
    firingrate[i] = 0.0;
  }
  /* TIME LOOP */
  size_t sizeOfInt = sizeof(int);
  size_t sizeOfDbl = sizeof(double);
  /* SETUP TIMER EVENTS ON DEVICE */
  hipEventCreate(&stop0); hipEventCreate(&start0);
  hipEventRecord(start0, 0);
  unsigned int spksE = 0, spksI = 0;
  FILE *fpIFR = fopen("instant_fr.csv", "w");
  int *histVec = NULL, *dev_histVec = NULL; /* for storing the post-synaptic neurons to be updated */
  int histVecIndx = 0;
  unsigned int histVecLength = N_NEURONS * (int)K;
  if((unsigned long long)K >= NE | (unsigned long long)K >= NI) {
    histVecLength = (unsigned int)(N_NEURONS * N_NEURONS);
  }
  cudaCheck(hipHostMalloc((void **)&histVec, histVecLength * sizeof(*histVec)));
  cudaCheck(hipMalloc((void **)&dev_histVec, histVecLength * sizeof(*dev_histVec)));
  test_xform xform; // defined in cuda_histogram.h
  test_sumfun sum;  // defined in cuda_histogram.h
  int *dev_histCountE = NULL, *histCountE = NULL, *dev_histCountI = NULL, *histCountI = NULL;;
  cudaCheck(hipMalloc((void **)&dev_histCountE, sizeof(int) * N_NEURONS));
  cudaCheck(hipHostMalloc((void **)&histCountE, sizeof(int) * N_NEURONS));
  cudaCheck(hipMalloc((void **)&dev_histCountI, sizeof(int) * N_NEURONS));
  cudaCheck(hipHostMalloc((void **)&histCountI, sizeof(int) * N_NEURONS));
  int tmp;
  char fileSuffix[128], filename[128];
  strcpy(filename, "currents");
  sprintf(fileSuffix, "_%1.1f_te%1.f_ti%1.f", ALPHA, TAU_SYNAP_E, TAU_SYNAP_I);
  strcat(filename, fileSuffix);
  FILE *fpCur = NULL;
  fpCur = fopen(strcat(filename, ".csv"), "w");
  /*printf("\n\n\n\n %d\n\n\n\n", sparseConVec[835584ULL]);*/
  for(k = 0; k < nSteps; ++k) { 
    /*    cudaCheck(hipMemsetAsync(dev_nEPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP * sizeOfInt, stream1));
	  cudaCheck(hipMemsetAsync(dev_nIPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP * sizeOfInt, stream1));*/
    /*    nSpksInPrevStep = 0;*/
    devPtrs.k = k;
    nSpksInPrevStep = 0;
    histVecIndx = 0;
    for(i = 0; i < N_NEURONS; ++i) {
      histCountI[i] = 0;
      histCountE[i] = 0;
    }

    rkdumbPretty<<<BlocksPerGrid, ThreadsPerBlock>>> (kernelParams, devPtrs);
    cudaCheckLastError("rk");
    if(k > 0) {
      /*      cudaCheck(hipMemcpy(host_IF_SPK, dev_IF_SPK_Ptr, N_NEURONS * sizeOfInt, hipMemcpyDeviceToHost));*/
      cudaCheck(hipMemcpyAsync(host_IF_SPK, dev_IF_SPK_Ptr, N_NEURONS * sizeOfInt, hipMemcpyDeviceToHost, stream1));
      cudaCheck(hipMemcpyAsync(host_isynap, synapticCurrent, N_I_SAVE_CUR * sizeOfDbl, hipMemcpyDeviceToHost, stream1));
    }
    cudaCheck(hipStreamSynchronize(stream1));
    /*instantaneous firing rate, rect non-overlapping window */
    for(i = 0; i < N_NEURONS; ++i) {
      if(host_IF_SPK[i]) {
	if(k * DT > DISCARDTIME) {
	  firingrate[i] += host_IF_SPK[i];
	}
	if(i < NE) {
	  spksE += 1;
	}
	else{
	  spksI += 1;
	}
	/*	    host_prevStepSpkIdx[i] = nSpksInPrevStep;
		    nSpksInPrevStep += 1;*/
      }
    }
    
    if(!(k%(int)(50.0/DT))) {
      fprintf(fpIFR, "%f %f \n", ((double)spksE) / (0.05 * (double)NE), ((double)spksI) / (0.05 * (double)NI));fflush(fpIFR);
      fprintf(stdout, "%f %f \n", ((double)spksE) / (0.05 * (double)NE), ((double)spksI) / (0.05 * (double)NI));
      spksE = 0; 
      spksI = 0;
    }
    /*-----------------------------------------------------------------------*/
    expDecay<<<BlocksPerGrid, ThreadsPerBlock>>>(dev_histCountE, dev_histCountI);
    cudaCheckLastError("exp");
    for(i = 0; i < NE; ++i) {
      if(host_IF_SPK[i]){
      nSpksInPrevStep += 1;
        for(int jj = 0; jj < nPostNeurons[i]; ++jj) {
          tmp = sparseConVec[idxVec[i] + jj];
          histVec[histVecIndx++] = tmp;
            /*          histVec[histVecIndx++] = sparseConVec[idxVec[i] + jj];*/
        }
      }
    }
    if(nSpksInPrevStep) {
      cudaCheck(hipMemcpy(dev_histVec, histVec, histVecIndx * sizeof(int), hipMemcpyHostToDevice));
      callHistogramKernel<histogram_atomic_inc, 1>(dev_histVec, xform, sum, 0, histVecIndx, 0, &histCountE[0], (int)N_NEURONS);
      /*      cudaCheck(hipMemcpy(dev_histCountE, histCountE, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));*/
      cudaCheck(hipMemcpyAsync(dev_histCountE, histCountE, N_NEURONS * sizeof(int), hipMemcpyHostToDevice, stream1));
    }
    
    histVecIndx = 0;
    nSpksInPrevStep = 0; 
    for(i = NE; i < N_NEURONS; ++i) {
      if(host_IF_SPK[i]){
        nSpksInPrevStep += 1;
        for(int jj = 0; jj < nPostNeurons[i]; ++jj) {
          histVec[histVecIndx++] = sparseConVec[idxVec[i] + jj];
        }
      }
    }
    
    if(nSpksInPrevStep) {
      cudaCheck(hipMemcpy(dev_histVec, histVec, histVecIndx * sizeof(int), hipMemcpyHostToDevice));
      callHistogramKernel<histogram_atomic_inc, 1>(dev_histVec, xform, sum, 0, histVecIndx, 0, &histCountI[0], (int)N_NEURONS);
      cudaCheckLastError("HIST");
      cudaCheck(hipMemcpy(dev_histCountI, histCountI, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
      /*      cudaCheck(hipMemcpyAsync(dev_histCountI, histCountI, N_NEURONS * sizeof(int), hipMemcpyHostToDevice, stream1));*/
    }


    /*    expDecay<<<BlocksPerGrid, ThreadsPerBlock>>>();*/

    /*computeConductance<<<BlocksPerGrid, ThreadsPerBlock>>>();*/
    cudaCheck(hipStreamSynchronize(stream1));

    /* SAVE CURRENT VALUES TO DISK  */
    for(int jj = 0; jj < N_I_SAVE_CUR; ++jj) {
      fprintf(fpCur, "%f ", host_isynap[jj]);
    }
    fprintf(fpCur, "\n");
    computeConductanceHist<<<(N_NEURONS + 512 - 1) / 512, 512>>>(dev_histCountE, dev_histCountI);
    cudaCheckLastError("g");
    computeIsynap<<<BlocksPerGrid, ThreadsPerBlock>>>(k*DT);
    cudaCheckLastError("isyp");
  }
  cudaCheck(hipHostFree(histVec));
  cudaCheck(hipFree(dev_histVec));
  cudaCheck(hipFree(dev_histCountE));
  cudaCheck(hipFree(dev_histCountI));
  cudaCheck(hipHostFree(histCountE));  
  cudaCheck(hipHostFree(histCountI));
  fclose(fpIFR);
  cudaCheck(hipStreamDestroy(stream1));
  cudaCheckLastError("rkdumb kernel failed");
  hipEventRecord(stop0, 0);
  hipEventSynchronize(stop0);
  printf(" Done ! \n");
  hipEventElapsedTime(&elapsedTime, start0, stop0);
  printf("\n elapsed time = %fms \n", elapsedTime);
  cudaCheck(hipEventDestroy(start0));
  cudaCheck(hipEventDestroy(stop0));
  /*==================== COPY RESULTS TO HOST =================================================*/
  cudaCheck(hipMemcpy(nSpks, dev_nSpks, sizeof(int), hipMemcpyDeviceToHost));
  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);
  cudaCheck(hipMemcpy(spkTimes, dev_spkTimes, MAX_SPKS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(spkNeuronIds, dev_spkNeuronIds, MAX_SPKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(host_time, dev_time, lastNStepsToStore * sizeof(double), hipMemcpyDeviceToHost));
  /*  cudaCheck(hipMemcpy(host_isynap, synapticCurrent, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));*/
  /*  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));*/
  double curE[N_CURRENT_STEPS_TO_STORE], curI[N_CURRENT_STEPS_TO_STORE], ibgCur[N_CURRENT_STEPS_TO_STORE], *dev_curE, *dev_curI, *dev_ibg, curIff[N_CURRENT_STEPS_TO_STORE], *dev_curiff;
  cudaCheck(hipGetSymbolAddress((void **)&dev_curE, glbCurE));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curI, glbCurI));
  cudaCheck(hipGetSymbolAddress((void **)&dev_ibg, dev_bgCur));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curiff, dev_iff));
  cudaCheck(hipMemcpy(curE, dev_curE, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curI, dev_curI, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(ibgCur, dev_ibg, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curIff, dev_curiff, N_CURRENT_STEPS_TO_STORE * sizeof(double), hipMemcpyDeviceToHost));
  printf("\n nSpks = %d\n", *nSpks);
  printf(" MAX SPKS stored on GPU = %d \n", MAX_SPKS); 
  printf("\n Simulation completed ! \n");
  fpElapsedTime = fopen("elapsedTime.csv", "a+");
  fprintf(fpElapsedTime, "%llu %f %d\n", N_NEURONS, elapsedTime, *nSpks);
  fclose(fpElapsedTime);
  /* ================= SAVE TO DISK =============================================================*/

  printf(" saving results to disk ... "); 
  fflush(stdout);
  //  char fileSuffix[128], filename[128];

  strcpy(filename, "spkTimes");
  sprintf(fileSuffix, "_xi%1.1f_theta%d_%.2f_%1.1f_cntrst%.1f_%d_tr%s", ETA_E, (int)theta_degrees, ALPHA, TAU_SYNAP_E, HOST_CONTRAST, (int)(tStop),filetag);
  strcat(filename, fileSuffix);
  fpSpkTimes = fopen(strcat(filename, ".csv"),"w");
  /*  fpSpkTimes = fopen("spkTimes.csv", "w");*/
  int totalNSpks = *nSpks;
  printf(" saving spikes ...");
  fflush(stdout);
  if(*nSpks > MAX_SPKS) {
    totalNSpks = MAX_SPKS;
    printf("\n ***** WARNING MAX_SPKS EXCEEDED limit of %d *****\n", MAX_SPKS);
  }
  if(IF_SAVE) {
      for(i = 1; i <= totalNSpks; ++i) {
        fprintf(fpSpkTimes, "%f;%f\n", spkTimes[i], (double)spkNeuronIds[i]);
      }
  }
  fclose(fpSpkTimes);
  printf("done\n");
  printf("computing firing rates ....");
  fflush(stdout);
  strcpy(filename, "firingrates");
  sprintf(fileSuffix, "_xi%1.1f_theta%d_%.2f_%1.1f_cntrst%.1f_%d_tr%s", ETA_E, (int)theta_degrees, ALPHA, TAU_SYNAP_E, HOST_CONTRAST, (int)(tStop),filetag);
  strcat(filename, fileSuffix);
  FILE *fpFiringrate = fopen(strcat(filename, ".csv"),"w");
  for(i = 0; i < N_NEURONS; ++i) {
    fprintf(fpFiringrate, "%f\n", firingrate[i] / ((tStop - DISCARDTIME) * 0.001));
  }
  fclose(fpFiringrate);
  printf("saving vm to disk ....");
  fflush(stdout);
  if(IF_SAVE) {
    //    char fileSuffix[128], filename[128];
    strcpy(filename, "vm");
    sprintf(fileSuffix, "_xi%1.1f_theta%d_%.2f_%1.1f_%d_tr%s", ETA_E, (int)theta_degrees, ALPHA, TAU_SYNAP_E, (int)(tStop), filetag);
    //sprintf(fileSuffix, "_%1.1f_%1.1f", ALPHA, TAU_SYNAP);
    strcat(filename, fileSuffix);
    fp = fopen(strcat(filename, ".csv"),"w");
    
    //    fp = fopen("vm.csv", "w");
    for(i = 0; i < lastNStepsToStore; ++i) {
      fprintf(fp, "%f ", host_time[i]);
      for(k = 0; k < N_NEURONS_TO_STORE; ++k) {
	/*	fprintf(fp, "%f %f ", vm[k + i *  N_NEURONS], host_isynap[k + i * N_NEURONS]);*/
        fprintf(fp, "%f ", vm[k + i *  N_NEURONS_TO_STORE]);
      }
      fprintf(fp, "\n");
    }
    printf("\n%d %d\n", i, k);
    fclose(fp);
    /*    FILE* fpCur = fopen("currents.csv", "w");*/
    FILE* fpCurbg = fopen("bgcur.csv", "w");
    for(i = 0; i < N_CURRENT_STEPS_TO_STORE; ++i) {
      fprintf(fpCur, "%f\n", ibgCur[i]);
    /*    fprintf(fpCur, "%f;%f;%f;%f\n", curE[i], curI[i], ibgCur[i], curIff[i]);*/
      /*      fprintf(fpCur, "%f\n", curIff[i]);*/
    }
    fclose(fpCurbg);
    fclose(fpCur);
    fpConMat = fopen("conMat.csv", "w");
    fpConMat = fopen("conVec.csv", "w");

    /*    for(i = 0; i < N_NEURONS; ++i) {
      for(k = 0; k < N_NEURONS; ++k) {
	fprintf(fpConMat, "%d", conVec[i *  N_NEURONS + k]);
      }
            fprintf(fpConMat, "\n");

      }*/
    fclose(fpConMat);
  }
  printf("done\n");
  /*================== CLEANUP ===================================================================*/
  free(firingrate);
  cudaCheck(hipHostFree(vm));
  cudaCheck(hipHostFree(host_time));
  /*  cudaCheck(hipHostFree(host_isynap));*/
  cudaCheck(hipHostFree(spkTimes));
  cudaCheck(hipHostFree(spkNeuronIds));
  cudaCheck(hipHostFree(nSpks));
  cudaCheck(hipFree(dev_vm));
  cudaCheck(hipFree(dev_time));
  /*  cudaCheck(hipFree(synapticCurrent));*/
  cudaCheck(hipFree(dev_spkNeuronIds));
  cudaCheck(hipFree(dev_spkTimes));
  cudaCheck(hipFree(dev_nSpks));
  cudaCheck(hipFree(devStates));
  cudaCheck(hipFree(dev_sparseVec));
      /*
  cudaCheck(hipFree(dev_idxVec));
  cudaCheck(hipFree(dev_nPostneuronsPtr));*/
  cudaCheck(hipHostFree(host_IF_SPK));
  cudaCheck(hipHostFree(host_prevStepSpkIdx));
  /*  hipDeviceReset()*/
  return EXIT_SUCCESS;
}

