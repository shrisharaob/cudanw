#include "hip/hip_runtime.h"
/* cuda network simulation 
   History :                    
    created: Shrisha
   Makefile included for build on CC=3.5
*/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "globalVars.h"
#include "aux.cu"

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}
void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

int main(int argc, char *argv[]) {
  double tStart = 0.0, tStop = 1000.0;
  double *spkTimes, *vm = NULL, host_theta = 0.0; /* *vstart; 500 time steps */
  int *nSpks, *spkNeuronIds, nSteps, i, k, lastNStepsToStore;
  double *dev_vm = NULL, *dev_spkTimes, *dev_time = NULL, *host_time;
  int *dev_conVec = NULL, *dev_nSpks, *dev_spkNeuronIds;
  FILE *fp, *fpConMat, *fpSpkTimes, *fpElapsedTime;
  double *host_isynap, *synapticCurrent;
  int *conVec;
  hiprandState *devStates, *devNormRandState;
  hipEvent_t start0, stop0;
  float elapsedTime;
  int *dev_sparseVec = NULL, *sparseConVec = NULL;
  int idxVec[N_NEURONS], nPostNeurons[N_NEURONS], *dev_idxVec = NULL, *dev_nPostneuronsPtr = NULL;
  int deviceId = 0;
  devPtr_t devPtrs;
  kernelParams_t kernelParams;
  int IF_SAVE = 1;
  /*PARSE INPUTS*/
  if(argc >1) {
    deviceId = atoi(argv[1]);
    if(argc > 2) {
      IF_SAVE = atoi(argv[2]);
    }
    if(argc > 3) {
      host_theta = atof(argv[3]);
    }
  }
  printf("\n Computing on GPU%d \n", deviceId);
  cudaCheck(hipSetDevice(deviceId));
  hipMemcpyToSymbol(HIP_SYMBOL(theta), &host_theta, sizeof(host_theta));
  /* ================= INITIALIZE ===============================================*/
  nSteps = (tStop - tStart) / DT;
  lastNStepsToStore = (int)floor(STORE_LAST_T_MILLISEC  / DT);
  //  nSteps = 800;
  printf("\n N  = %d \n NE = %d \n NI = %d \n K  = %d \n tStop = %3.2f seconds nSteps = %d\n\n", N_NEURONS, NE, NI, (int)K, tStop / 1000.0, nSteps);
  printf(" theta = %2.1f\n", host_theta);
  
  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = 128;
  int BlocksPerGrid = (N_NEURONS + ThreadsPerBlock - 1) / ThreadsPerBlock;
  printf("Threads per block : %d, Blocks per grid : %d \n", ThreadsPerBlock, BlocksPerGrid);
  /*INITIALIZE RND GENERATORS FOR ibf & iff */
  setupBGCurGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  setupIFFRndGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  /*Generate frozen FF input approximat*/
  cudaCheck(hipMalloc((void **)&devStates,  N_NEURONS * sizeof(hiprandState)));
  cudaCheck(hipMalloc((void **)&devNormRandState, N_NEURONS * sizeof(hiprandState)));
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, time(NULL));
  AuxRffTotal<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, devStates);
  cudaCheck(hipFree(devNormRandState));
  /* gENERATE CONNECTION MATRIX */
  /*  cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));*/
  /*  cudaCheck(hipHostMalloc((void **)&conVec, N_NEURONS * N_NEURONS * sizeof(int)));  */
  /*  cudaCheck(hipMemset(dev_conVec, 0, N_NEURONS * N_NEURONS * sizeof(int)));
  printf("\n launching rand generator setup kernel\n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  printf("\n launching connection matrix geneting kernel with seed %ld ...", time(NULL));
  fflush(stdout);
  kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVec);
  printf(" Done! \n");
  cudaCheck(hipMemcpy(conVec, dev_conVec, N_NEURONS * N_NEURONS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipFree(dev_conVec));*/
  /*  printf("reading convec.dat..."); fflush(stdout);
  FILE *fpConVecFile = fopen("conVec.dat", "rb");
  fread(conVec, sizeof(*conVec), N_NEURONS * N_NEURONS, fpConVecFile);
  fclose(fpConVecFile);
  printf("done ...\n");*/
  /* SPARSIFY */
  /*  conVec[0] = 0; conVec[1] = 0; conVec[2] = 1;conVec[3] = 0;*/
  /*conVec[4] = 0;conVec[5] = 1;conVec[6] = 1;conVec[7] = 1;
  conVec[8] = 1;*/ /*conVec[9] = 0;*/
  /*conVec[10] = 0;conVec[11] = 1;
    conVec[12]= 0;conVec[13] = 0;conVec[14] = 0;conVec[15] = 0;*/
  cudaCheck(hipGetSymbolAddress((void **)&dev_sparseVec, dev_sparseConVec));
  cudaCheck(hipGetSymbolAddress((void **)&dev_idxVec, dev_sparseIdx));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nPostneuronsPtr, dev_nPostNeurons));
  cudaCheck(hipHostMalloc((void **)&sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int)));
  /*  cudaCheck(hipMalloc((void **)&dev_sparseVec, N_NEURONS * ((int)2 * K + 1)* sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_idxVec, N_NEURONS * sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_nPostneuronsPtr, N_NEURONS * sizeof(int)));*/
  /*  GenSparseMat(conVec, N_NEURONS, N_NEURONS, sparseConVec, idxVec, nPostNeurons);*/
  FILE *fpSparseConVec, *fpIdxVec, *fpNpostNeurons;
  fpSparseConVec = fopen("sparseConVec.dat", "rb");
  fpIdxVec = fopen("idxVec.dat", "rb");
  fpNpostNeurons = fopen("nPostNeurons.dat", "rb");
  fread(sparseConVec, sizeof(*sparseConVec), N_NEURONS * (2 * K + 1), fpSparseConVec);
  fread(idxVec, sizeof(*idxVec), N_NEURONS, fpIdxVec);
  fread(nPostNeurons, sizeof(*nPostNeurons), N_NEURONS, fpNpostNeurons);
  fclose(fpSparseConVec);
  fclose(fpIdxVec);
  fclose(fpNpostNeurons);
  /*
    for(i = 0; i < N_NEURONS; ++i) {
      printf("neuron %d projects to : ", i);
      for(int j = 0; j < nPostNeurons[i]; ++j) {
	printf("%d ", sparseConVec[idxVec[i] + j]);
      }
      printf("\n");
    }
  */

  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVec, idxVec, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtr, nPostNeurons, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
 /* ================= ALLOCATE PAGELOCKED MEMORY ON HOST =========================*/
  cudaCheck(hipHostMalloc((void **)&spkTimes, MAX_SPKS  * sizeof(*spkTimes)));
  cudaCheck(hipHostMalloc((void **)&host_isynap, lastNStepsToStore * N_NEURONS * sizeof(*host_isynap)));
  cudaCheck(hipHostMalloc((void **)&vm,  lastNStepsToStore * N_NEURONS * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&host_time,  lastNStepsToStore * N_NEURONS * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&nSpks, sizeof(*nSpks)));
  cudaCheck(hipHostMalloc((void **)&spkNeuronIds, MAX_SPKS * sizeof(*spkNeuronIds)));
  /* ================= ALLOCATE GLOBAL MEMORY ON DEVICE ===========================*/
  /*cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));*/
  cudaCheck(hipMalloc((void **)&dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_time, lastNStepsToStore * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&synapticCurrent, lastNStepsToStore * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_spkTimes, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMalloc((void **)&dev_nSpks, sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_spkNeuronIds, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  cudaCheck(hipMemset(dev_spkTimes, 0, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMemset(dev_spkNeuronIds, 0.0f, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  printf(" GPU memory allocation successful ! \n ");
  devPtrs.dev_conVec = dev_conVec;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_vm = dev_vm;
  devPtrs.dev_nSpks = dev_nSpks;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_spkTimes = dev_spkTimes;
  devPtrs.synapticCurrent = synapticCurrent;
  devPtrs.devStates = devStates;
  /*  devPtrs.dev_sparseConVec = dev_sparseVec;
  devPtrs.dev_nPostNeurons = dev_nPostneuronsPtr;
  devPtrs.dev_sparseIdx = dev_idxVec;*/
  devPtrs.dev_time = dev_time;
  *nSpks = 0;
  cudaCheck(hipMemcpy(dev_nSpks, nSpks, sizeof(int), hipMemcpyHostToDevice));
  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);
  /* INTEGRATE ODEs ON GPU*/
  /* invoke device on this block/thread grid */
  kernelParams.nSteps = nSteps;
  kernelParams.tStop = tStop;
  kernelParams.tStart = tStart;
  printf("\n launching Simulation kernel ...");
  fflush(stdout);
  /* SETUP TIMER EVENTS ON DEVICE */
  hipEventCreate(&stop0); hipEventCreate(&start0);
  hipEventRecord(start0, 0);
  /* TIME LOOP */
  for(k = 0; k < nSteps; ++k) { 
    devPtrs.k = k;
    rkdumbPretty<<<BlocksPerGrid, ThreadsPerBlock>>> (kernelParams, devPtrs);
    expDecay<<<BlocksPerGrid, ThreadsPerBlock>>>();
    computeConductance<<<BlocksPerGrid, ThreadsPerBlock>>>();
    computeIsynap<<<BlocksPerGrid, ThreadsPerBlock>>>();
  }
  /* RECORD COMPUTE TIME */
  cudaCheckLastError("rkdumb kernel failed");
  hipEventRecord(stop0, 0);
  hipEventSynchronize(stop0);
  printf(" Done ! \n");
  hipEventElapsedTime(&elapsedTime, start0, stop0);
  printf("\n elapsed time = %fms \n", elapsedTime);
  cudaCheck(hipEventDestroy(start0));
  cudaCheck(hipEventDestroy(stop0));
  /*==================== COPY RESULTS TO HOST =================================================*/
  cudaCheck(hipMemcpy(nSpks, dev_nSpks, sizeof(int), hipMemcpyDeviceToHost));
  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);
  cudaCheck(hipMemcpy(spkTimes, dev_spkTimes, MAX_SPKS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(spkNeuronIds, dev_spkNeuronIds, MAX_SPKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(host_time, dev_time, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(host_isynap, synapticCurrent, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  double curE[5 * 4000], curI[5 * 4000], ibgCur[5 * 4000], *dev_curE, *dev_curI, *dev_ibg, curIff[5000], *dev_curiff;
  cudaCheck(hipGetSymbolAddress((void **)&dev_curE, glbCurE));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curI, glbCurI));
  cudaCheck(hipGetSymbolAddress((void **)&dev_ibg, dev_bgCur));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curiff, dev_iff));
  cudaCheck(hipMemcpy(curE, dev_curE, 5 * 4000 * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curI, dev_curI, 5 * 4000 * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(ibgCur, dev_ibg, 5 * 4000 * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curIff, dev_curiff, 5000 * sizeof(double), hipMemcpyDeviceToHost));
  printf("\n nSpks = %d\n", *nSpks);
  printf(" MAX SPKS stored on GPU = %d \n", MAX_SPKS); 
  printf("\n Simulation completed ! \n");
  fpElapsedTime = fopen("elapsedTime.csv", "a+");
  fprintf(fpElapsedTime, "%d %f %d\n", N_NEURONS, elapsedTime, *nSpks);
  fclose(fpElapsedTime);
  /* ================= SAVE TO DISK =============================================================*/
  if(IF_SAVE) {  
    printf(" saving results to disk ... "); 
    fflush(stdout);
    fpSpkTimes = fopen("spkTimes.csv", "w");
    int totalNSpks = *nSpks;
    printf(" saving spikes ...");
    fflush(stdout);
    if(*nSpks > MAX_SPKS) {
      totalNSpks = MAX_SPKS;
      printf("\n ***** WARNING MAX_SPKS EXCEEDED limit of %d *****\n", MAX_SPKS);
    }
    for(i = 1; i <= totalNSpks; ++i) {
      fprintf(fpSpkTimes, "%f;%f\n", spkTimes[i], (double)spkNeuronIds[i]);
    }
    fclose(fpSpkTimes);
    printf("done\n");
    fp = fopen("vm.csv", "w");
    for(i = 0; i < lastNStepsToStore; ++i) {
      fprintf(fp, "%f ", host_time[i]);
      for(k = 0; k < N_NEURONS; ++k) {
	/*	fprintf(fp, "%f %f ", vm[k + i *  N_NEURONS], host_isynap[k + i * N_NEURONS]);*/
	fprintf(fp, "%f ", vm[k + i *  N_NEURONS]);
      }
      fprintf(fp, "\n");
    }
    fclose(fp);
    FILE* fpCur = fopen("currents.csv", "w");
    for(i = 0; i < 5000; ++i) {
      fprintf(fpCur, "%f;%f;%f;%f\n", curE[i], curI[i], ibgCur[i], curIff[i]);
    }
    fclose(fpCur);
    fpConMat = fopen("conMat.csv", "w");
    fpConMat = fopen("conVec.csv", "w");
    /*	  fwrite(conVec, size(int), N_NEURONS * N_NEURONS, fpConMat);*/
    /*    for(i = 0; i < N_NEURONS; ++i) {
      for(k = 0; k < N_NEURONS; ++k) {
	fprintf(fpConMat, "%d", conVec[i *  N_NEURONS + k]);
      }
            fprintf(fpConMat, "\n");
	    }*/
    fclose(fpConMat);
  }
  /*================== CLEANUP ===================================================================*/
  cudaCheck(hipHostFree(vm));
  cudaCheck(hipHostFree(host_time));
  cudaCheck(hipHostFree(host_isynap));
  cudaCheck(hipHostFree(spkTimes));
  cudaCheck(hipHostFree(spkNeuronIds));
  cudaCheck(hipHostFree(nSpks));
  cudaCheck(hipFree(dev_vm));
  cudaCheck(hipFree(dev_time));
  cudaCheck(hipFree(synapticCurrent));
  cudaCheck(hipFree(dev_spkNeuronIds));
  cudaCheck(hipFree(dev_spkTimes));
  cudaCheck(hipFree(dev_nSpks));
  cudaCheck(hipFree(devStates));
  /*  cudaCheck(hipFree(dev_sparseVec));
  cudaCheck(hipFree(dev_idxVec));
  cudaCheck(hipFree(dev_nPostneuronsPtr));*/
  return EXIT_SUCCESS;
}

