#include "hip/hip_runtime.h"
/* cuda network simulation 
   History :                    
    created: Shrisha
   Makefile included for build on CC=3.5
*/
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "globalVars.h"
#include "aux.cu"

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}
void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

int main(int argc, char *argv[]) {
  double tStart = 0.0, tStop = 100.0;
  double *spkTimes, *vm = NULL, host_theta = 0.0; /* *vstart; 500 time steps */
  int *nSpks, *spkNeuronIds, nSteps, i, k, lastNStepsToStore;
  double *dev_vm = NULL, *dev_spkTimes, *dev_time = NULL, *host_time;
  int *dev_conVec, *dev_nSpks, *dev_spkNeuronIds;
  FILE *fp, *fpConMat, *fpSpkTimes, *fpElapsedTime;
  double *host_isynap, *synapticCurrent;
  int *conVec;
  hiprandState *devStates, *devNormRandState;
  hipEvent_t start0, stop0;
  float elapsedTime;
  int *dev_sparseVec = NULL, *sparseConVec = NULL;
  int idxVec[N_NEURONS], nPostNeurons[N_NEURONS], *dev_idxVec = NULL, *dev_nPostneuronsPtr = NULL;
  int deviceId = 0;
  devPtr_t devPtrs;
  kernelParams_t kernelParams;
  int IF_SAVE = 1;
  hipStream_t stream1;
  cudaCheck(hipStreamCreate(&stream1));

  /*PARSE INPUTS*/
  if(argc >1) {
    deviceId = atoi(argv[1]);
    if(argc > 2) {
      IF_SAVE = atoi(argv[2]);
    }
    if(argc > 3) {
      host_theta = atof(argv[3]);
    }
  }
  printf("\n Computing on GPU%d \n", deviceId);
  cudaCheck(hipSetDevice(deviceId));
  hipMemcpyToSymbol(HIP_SYMBOL(theta), &host_theta, sizeof(host_theta));
  /* ================= INITIALIZE ===============================================*/
  nSteps = (tStop - tStart) / DT;
  lastNStepsToStore = (int)floor(STORE_LAST_T_MILLISEC  / DT);
  //  nSteps = 800;
  printf("\n N  = %d \n NE = %d \n NI = %d \n K  = %d \n tStop = %3.2f seconds nSteps = %d\n\n", N_NEURONS, NE, NI, (int)K, tStop / 1000.0, nSteps);
  printf(" theta = %2.1f\n", host_theta);
  /* ================== SETUP TIMER EVENTS ON DEVICE ==============================*/
  hipEventCreate(&stop0); hipEventCreate(&start0);
  hipEventRecord(start0, 0);
  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = 128;
  int BlocksPerGrid = (N_NEURONS + ThreadsPerBlock - 1) / ThreadsPerBlock;
  printf("Threads per block : %d, Blocks per grid : %d \n", ThreadsPerBlock, BlocksPerGrid);
  /*INITIALIZE RND GENERATORS FOR ibf & iff */
  setupBGCurGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  setupIFFRndGenerator<<<BlocksPerGrid, ThreadsPerBlock>>>(time(NULL));
  /*Generate frozen FF input approximat*/
  cudaCheck(hipMalloc((void **)&devStates,  N_NEURONS * sizeof(hiprandState)));
  cudaCheck(hipMalloc((void **)&devNormRandState, N_NEURONS * sizeof(hiprandState)));
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, time(NULL));
  AuxRffTotal<<<BlocksPerGrid, ThreadsPerBlock>>>(devNormRandState, devStates);
  cudaCheck(hipFree(devNormRandState));
  /* gENERATE CONNECTION MATRIX */
  cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));
  cudaCheck(hipHostMalloc((void **)&conVec, N_NEURONS * N_NEURONS * sizeof(int)));  
  cudaCheck(hipMemset(dev_conVec, 0, N_NEURONS * N_NEURONS * sizeof(int)));
  printf("\n launching rand generator setup kernel\n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  printf("\n launching connection matrix geneting kernel with seed %ld ...", time(NULL));
  fflush(stdout);
  kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVec);
  printf(" Done! \n");
  cudaCheck(hipMemcpy(conVec, dev_conVec, N_NEURONS * N_NEURONS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipFree(dev_conVec));
  /* SPARSIFY */
  conVec[0] = 0; conVec[1] = 1; conVec[2] = 1;conVec[3] = 0;
  /*conVec[4] = 0;conVec[5] = 1;conVec[6] = 1;conVec[7] = 1;
  conVec[8] = 1;*/ /*conVec[9] = 0;*/
  /*conVec[10] = 0;conVec[11] = 1;
    conVec[12]= 0;conVec[13] = 0;conVec[14] = 0;conVec[15] = 0;*/
  cudaCheck(hipGetSymbolAddress((void **)&dev_sparseVec, dev_sparseConVec));
  cudaCheck(hipGetSymbolAddress((void **)&dev_idxVec, dev_sparseIdx));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nPostneuronsPtr, dev_nPostNeurons));
  cudaCheck(hipHostMalloc((void **)&sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int)));  
  /*  cudaCheck(hipMalloc((void **)&dev_sparseVec, N_NEURONS * ((int)2 * K + 1)* sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_idxVec, N_NEURONS * sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_nPostneuronsPtr, N_NEURONS * sizeof(int)));*/
  GenSparseMat(conVec, N_NEURONS, N_NEURONS, sparseConVec, idxVec, nPostNeurons);
  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVec, idxVec, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtr, nPostNeurons, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
 /* ================= ALLOCATE PAGELOCKED MEMORY ON HOST =========================*/
  cudaCheck(hipHostMalloc((void **)&spkTimes, MAX_SPKS  * sizeof(*spkTimes)));
  cudaCheck(hipHostMalloc((void **)&host_isynap, lastNStepsToStore * N_NEURONS * sizeof(*host_isynap)));
  cudaCheck(hipHostMalloc((void **)&vm,  lastNStepsToStore * N_NEURONS * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&host_time,  lastNStepsToStore * N_NEURONS * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&nSpks, sizeof(*nSpks)));
  cudaCheck(hipHostMalloc((void **)&spkNeuronIds, MAX_SPKS * sizeof(*spkNeuronIds)));
  /* ================= ALLOCATE GLOBAL MEMORY ON DEVICE ===========================*/
  /*cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));*/
  cudaCheck(hipMalloc((void **)&dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_time, lastNStepsToStore * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&synapticCurrent, lastNStepsToStore * N_NEURONS * sizeof(double)));
  cudaCheck(hipMalloc((void **)&dev_spkTimes, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMalloc((void **)&dev_nSpks, sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_spkNeuronIds, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  cudaCheck(hipMemset(dev_spkTimes, 0, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMemset(dev_spkNeuronIds, 0.0f, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
  printf(" GPU memory allocation successful ! \n ");
  devPtrs.dev_conVec = dev_conVec;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_vm = dev_vm;
  devPtrs.dev_nSpks = dev_nSpks;
  devPtrs.dev_spkNeuronIds = dev_spkNeuronIds;
  devPtrs.dev_spkTimes = dev_spkTimes;
  devPtrs.synapticCurrent = synapticCurrent;
  devPtrs.devStates = devStates;
  /*  devPtrs.dev_sparseConVec = dev_sparseVec;
  devPtrs.dev_nPostNeurons = dev_nPostneuronsPtr;
  devPtrs.dev_sparseIdx = dev_idxVec;*/
  devPtrs.dev_time = dev_time;
  *nSpks = 0;
  cudaCheck(hipMemcpy(dev_nSpks, nSpks, sizeof(int), hipMemcpyHostToDevice));
  /*  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);*/
  /*===================== GENERATE CONNECTION MATRIX ====================================*/
  /*cudaCheck(hipMemset(dev_conVec, 0, N_NEURONS * N_NEURONS * sizeof(int)));
  printf("\n launching rand generator setup kernel\n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  printf("\n launching connection matrix geneting kernel with seed %ld ...", time(NULL));
  fflush(stdout);
  kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVec);
  printf(" Done! \n");
  cudaCheck(hipMemcpy(conVec, dev_conVec, N_NEURONS * N_NEURONS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipFree(dev_conVec));
  GenSparseMat(conVec, N_NEURONS, N_NEURONS, sparseConVec, idxVec, nPostNeurons);
  cudaCheck(hipMemcpy(dev_sparseVec, sparseConVec, N_NEURONS * (2 * K + 1) * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_idxVec, idxVec, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dev_nPostneuronsPtr, nPostNeurons, N_NEURONS * sizeof(int), hipMemcpyHostToDevice));*/
  /* ==================== INTEGRATE ODEs ON GPU ==========================================*/
    /* invoke device on this block/thread grid */
  kernelParams.nSteps = nSteps;
  kernelParams.tStop = tStop;
  kernelParams.tStart = tStart;
  printf("\n launching Simulation kernel ...");
  fflush(stdout);
  int *dev_IF_SPK_Ptr = NULL, *dev_prevStepSpkIdxPtr = NULL, *host_IF_SPK = NULL, *host_prevStepSpkIdx = NULL, nSpksInPrevStep, *dev_nEPtr = NULL, *dev_nIPtr = NULL;
  cudaCheck(hipHostMalloc((void **)&host_IF_SPK, N_NEURONS * sizeof(int)));
  cudaCheck(hipHostMalloc((void **)&host_prevStepSpkIdx, N_NEURONS * sizeof(int)));
  cudaCheck(hipGetSymbolAddress((void **)&dev_IF_SPK_Ptr, dev_IF_SPK));
  cudaCheck(hipGetSymbolAddress((void **)&dev_prevStepSpkIdxPtr, dev_prevStepSpkIdx));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nEPtr, dev_ESpkCountMat));
  cudaCheck(hipGetSymbolAddress((void **)&dev_nIPtr, dev_ISpkCountMat));
  /* TIME LOOP */
  size_t sizeOfInt = sizeof(int);
  for(k = 0; k < nSteps; ++k) { 
    /*    cudaCheck(hipMemsetAsync(dev_nEPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP * sizeOfInt, stream1));
	  cudaCheck(hipMemsetAsync(dev_nIPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP * sizeOfInt, stream1));*/
    nSpksInPrevStep = 0;
    devPtrs.k = k;
    rkdumbPretty<<<BlocksPerGrid, ThreadsPerBlock>>> (kernelParams, devPtrs);
    cudaCheckLastError("rk");
    cudaCheck(hipMemcpy(host_IF_SPK, dev_IF_SPK_Ptr, N_NEURONS * sizeOfInt, hipMemcpyDeviceToHost));
    for(i = 0; i < N_NEURONS; ++i) {
      if(host_IF_SPK[i]) {
	host_prevStepSpkIdx[i] = nSpksInPrevStep;
        nSpksInPrevStep += 1;
      }
    }
    if(nSpksInPrevStep > N_SPKS_IN_PREV_STEP) { 
      printf("\nExceeded N_SPKS_IN_PREV_STEP ! nSpksInPrevStep = %d \n", nSpksInPrevStep); 
      exit(-1);
    }
    cudaCheck(hipMemcpy(dev_prevStepSpkIdxPtr, host_prevStepSpkIdx, N_NEURONS * sizeOfInt, hipMemcpyHostToDevice));
    expDecay<<<BlocksPerGrid, ThreadsPerBlock>>>();
    /*    cudaCheck(hipMemset(dev_nEPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP));
	  cudaCheck(hipMemset(dev_nIPtr, 0, N_NEURONS * N_SPKS_IN_PREV_STEP));*/
    /*    hipStreamSynchronize(stream1);*/
    /*    computeG_Optimal<<<BlocksPerGrid, ThreadsPerBlock>>>();
	  spkSum<<<BlocksPerGrid, ThreadsPerBlock>>>(nSpksInPrevStep);*/
    computeConductance<<<BlocksPerGrid, ThreadsPerBlock>>>();
    computeIsynap<<<BlocksPerGrid, ThreadsPerBlock>>>(k*DT);
  }
  cudaCheck(hipStreamDestroy(stream1));
  cudaCheckLastError("rkdumb kernel failed");
  /*==================== COPY RESULTS TO HOST =================================================*/
  cudaCheck(hipMemcpy(nSpks, dev_nSpks, sizeof(int), hipMemcpyDeviceToHost));
  printf("devspk ptrs: %p %p \n", dev_spkTimes, dev_spkNeuronIds);
  cudaCheck(hipMemcpy(spkTimes, dev_spkTimes, MAX_SPKS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(spkNeuronIds, dev_spkNeuronIds, MAX_SPKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(host_time, dev_time, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(host_isynap, synapticCurrent, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(vm, dev_vm, lastNStepsToStore * N_NEURONS * sizeof(double), hipMemcpyDeviceToHost));
  double curE[5 * 4000], curI[5 * 4000], ibgCur[5 * 4000], *dev_curE, *dev_curI, *dev_ibg, curIff[5000], *dev_curiff;
  cudaCheck(hipGetSymbolAddress((void **)&dev_curE, glbCurE));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curI, glbCurI));
  cudaCheck(hipGetSymbolAddress((void **)&dev_ibg, dev_bgCur));
  cudaCheck(hipGetSymbolAddress((void **)&dev_curiff, dev_iff));
  cudaCheck(hipMemcpy(curE, dev_curE, 5 * 4000 * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curI, dev_curI, 5 * 4000 * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(ibgCur, dev_ibg, 5 * 4000 * sizeof(double), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(curIff, dev_curiff, 5000 * sizeof(double), hipMemcpyDeviceToHost));
  /* ================= RECORD COMPUTE TIME ====================================================*/
  hipEventRecord(stop0, 0);
  hipEventSynchronize(stop0);
  printf(" Done ! \n");
  hipEventElapsedTime(&elapsedTime, start0, stop0);
  printf("\n elapsed time = %fms \n", elapsedTime);
  cudaCheck(hipEventDestroy(start0));
  cudaCheck(hipEventDestroy(stop0));
  printf("\n nSpks = %d\n", *nSpks);
  printf(" MAX SPKS stored on GPU = %d \n", MAX_SPKS); 
  printf("\n Simulation completed ! \n");
  fpElapsedTime = fopen("elapsedTime.csv", "a+");
  fprintf(fpElapsedTime, "%d %f %d\n", N_NEURONS, elapsedTime, *nSpks);
  fclose(fpElapsedTime);
  /* ================= SAVE TO DISK =============================================================*/
  if(IF_SAVE) {  
    printf(" saving results to disk ... "); 
    fflush(stdout);
    fpSpkTimes = fopen("spkTimes.csv", "w");
    int totalNSpks = *nSpks;
    printf(" saving spikes ...");
    fflush(stdout);
    if(*nSpks > MAX_SPKS) {
      totalNSpks = MAX_SPKS;
      printf("\n ***** WARNING MAX_SPKS EXCEEDED limit of %d *****\n", MAX_SPKS);
    }
    for(i = 1; i <= totalNSpks; ++i) {
      fprintf(fpSpkTimes, "%f;%f\n", spkTimes[i], (double)spkNeuronIds[i]);
    }
    printf("Done!\n");
    fclose(fpSpkTimes);

    fp = fopen("vm.csv", "w");
    for(i = 0; i < lastNStepsToStore; ++i) {
      fprintf(fp, "%f ", host_time[i]);
      for(k = 0; k < N_NEURONS; ++k) {
	/*	fprintf(fp, "%f %f ", vm[k + i *  N_NEURONS], host_isynap[k + i * N_NEURONS]);*/
	fprintf(fp, "%f ", vm[k + i *  N_NEURONS]);
      }
      fprintf(fp, "\n");
    }
    fclose(fp);
    FILE* fpCur = fopen("currents.csv", "w");
    for(i = 0; i < 5000; ++i) {
      fprintf(fpCur, "%f;%f;%f;%f\n", curE[i], curI[i], ibgCur[i], curIff[i]);
    }
    fclose(fpCur);
    /*fpConMat = fopen("conMat.csv", "w");*/
    fpConMat = fopen("conVec.csv", "w");
    /*    for(i = 0; i < N_NEURONS; ++i) {
      for(k = 0; k < N_NEURONS; ++k) {
	fprintf(fpConMat, "%d", conVec[i *  N_NEURONS + k]);
      }
            fprintf(fpConMat, "\n");
      }*/
    fclose(fpConMat);
  }
  /*================== CLEANUP ===================================================================*/
  cudaCheck(hipHostFree(vm));
  cudaCheck(hipHostFree(host_time));
  cudaCheck(hipHostFree(host_isynap));
  cudaCheck(hipHostFree(spkTimes));
  cudaCheck(hipHostFree(spkNeuronIds));
  cudaCheck(hipHostFree(nSpks));
  cudaCheck(hipFree(dev_vm));
  cudaCheck(hipFree(dev_time));
  cudaCheck(hipFree(synapticCurrent));
  cudaCheck(hipFree(dev_spkNeuronIds));
  cudaCheck(hipFree(dev_spkTimes));
  cudaCheck(hipFree(dev_nSpks));
  cudaCheck(hipFree(devStates));
  /*  cudaCheck(hipFree(dev_sparseVec));
  cudaCheck(hipFree(dev_idxVec));
  cudaCheck(hipFree(dev_nPostneuronsPtr));*/
  cudaCheck(hipHostFree(host_IF_SPK));
  cudaCheck(hipHostFree(host_prevStepSpkIdx));
  return EXIT_SUCCESS;
}

