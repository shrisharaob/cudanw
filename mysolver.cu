#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "globalVars.h"
#include "aux.cu"

void __cudaCheck(hipError_t err, const char* file, const int line);
#define cudaCheck(err) __cudaCheck (err, __FILE__, __LINE__)

void __cudaCheckLastError(const char* errorMessage, const char* file, const int line);
#define cudaCheckLastError(msg) __cudaCheckLastError (msg, __FILE__, __LINE__)

void __cudaCheck(hipError_t err, const char *file, const int line)
{
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
      file, line, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

void __cudaCheckLastError(const char *errorMessage, const char *file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

int main() {
  float tStart = 0.0, tStop = 100.0;
  float *spkTimes, *vm = NULL;// *vstart; // 500 time steps
  int *nSpks, *spkNeuronIds, nSteps, i, k;
  float *dev_vm = NULL, *dev_spkTimes;
  int *dev_conVec, *dev_nSpks, *dev_spkNeuronIds;
  FILE *fp, *fpConMat, *fpSpkTimes;
  float *host_isynap, *dev_isynap;
  int *conVec;
  hiprandState *devStates;
  hipEvent_t start0, stop0;
  float elapsedTime;
  hipError_t devErr;
  /* ================= INITIALIZE ===============================================*/
  nSteps = (tStop - tStart) / DT;
  //  nSteps = 800;
  printf("\n N  = %d \n NE = %d \n NI = %d \n K  = %d \n nSteps = %d\n\n", N_NEURONS, NE, NI, (int)K, nSteps);
  /* ================== SETUP TIMER EVENTS ON DEVICE ==============================*/
  hipEventCreate(&stop0); hipEventCreate(&start0);
  hipEventRecord(start0, 0);
  /* choose 256 threads per block for high occupancy */
  int ThreadsPerBlock = 128;
  int BlocksPerGrid = (N_NEURONS + ThreadsPerBlock - 1) / ThreadsPerBlock;
  /* ================= ALLOCATE PAGELOCKED MEMORY ON HOST =========================*/
  cudaCheck(hipHostMalloc((void **)&spkTimes, MAX_SPKS  * sizeof(*spkTimes)));
  cudaCheck(hipHostMalloc((void **)&host_isynap, nSteps * N_NEURONS * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&vm, nSteps * N_NEURONS * sizeof(*vm)));
  cudaCheck(hipHostMalloc((void **)&nSpks, sizeof(*nSpks)));
  cudaCheck(hipHostMalloc((void **)&spkNeuronIds, MAX_SPKS * sizeof(spkNeuronIds)));
  /*cudaCheck(hipHostMalloc((void **)&vstart, N_STATEVARS * N_NEURONS * sizeof(float)));*/
  cudaCheck(hipHostMalloc((void **)&conVec, N_NEURONS * N_NEURONS * sizeof(int)));
  /* ================= ALLOCATE GLOBAL MEMORY ON DEVICE ===========================*/
  cudaCheck(hipMalloc((void **)&dev_conVec, N_NEURONS * N_NEURONS * sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_vm, nSteps * N_NEURONS * sizeof(float)));
  cudaCheck(hipMalloc((void **)&dev_isynap, nSteps * N_NEURONS * sizeof(float)));
  cudaCheck(hipMalloc((void **)&dev_spkTimes, MAX_SPKS * sizeof(*dev_spkTimes)));
  cudaCheck(hipMalloc((void **)&dev_nSpks, sizeof(int)));
  cudaCheck(hipMalloc((void **)&dev_spkNeuronIds, MAX_SPKS * sizeof(*dev_spkNeuronIds)));
 /*cudaCheck(hipMalloc((void **)&dev_vstart, N_STATEVARS * N_NEURONS * sizeof(*dev_vstart)));*/
  cudaCheck(hipMalloc((void **)&devStates,  N_NEURONS * sizeof(hiprandState)));
  printf(" GPU memory allocation successful ! \n ");
  // printf("&dev_conVec = %p \n", dev_conVec);
  // printf("&devStates = %p\n", devStates);
  // printf("&conVec = %p\n", conVec); 
  // for(kNeuron = 0; kNeuron < N_Neurons; ++kNeuron) {
  //   int clmNo =  kNeuron * N_STATEVARS;
  //   vstart[0 + clmNo] = -50; //-70 +  40 * CudaURand(); // Vm(0) ~ U(-70, -30)
  //   vstart[1 + clmNo] = 0.3176;
  //   vstart[2 + clmNo] = 0.1;
  //   vstart[3 + clmNo] = 0.5961;
  // }
  /* cudaCheck(hipMemcpy(dev_vstart, vstart, N_STATEVARS * N_Neurons * sizeof(float), hipMemcpyHostToDevice));*/
  
  *nSpks = 0;
  cudaCheck(hipMemcpy(dev_nSpks, nSpks, sizeof(int), hipMemcpyHostToDevice));
  /*===================== GENERATE CONNECTION MATRIX ====================================*/
  cudaCheck(hipMemset(dev_conVec, 0, N_NEURONS * N_NEURONS * sizeof(int)));
  printf("\n launching rand generator setup kernel\n");
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL));
  printf("\n launching connection matrix geneting kernel with seed %ld ...", time(NULL));
  fflush(stdout);
  kernelGenConMat<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, dev_conVec);
  printf(" Done! \n");

  cudaCheck(hipMemcpy(conVec, dev_conVec, N_NEURONS * N_NEURONS * sizeof(int), hipMemcpyDeviceToHost));
  /* ==================== INTEGRATE ODEs ON GPU ==========================================*/
    /* invoke device on this block/thread grid */
  printf("\n launching Simulation kernel ...");
  fflush(stdout);
  setup_kernel<<<BlocksPerGrid, ThreadsPerBlock>>>(devStates, time(NULL) + 23);
  rkdumb <<<BlocksPerGrid,ThreadsPerBlock>>> (tStart, tStop, nSteps, dev_nSpks, dev_spkTimes, dev_spkNeuronIds, dev_vm, dev_conVec, dev_isynap, devStates);
  cudaCheckLastError("rkdumb kernel failed");
  /*==================== COPY RESULTS TO HOST =================================================*/
  cudaCheck(hipMemcpy(nSpks, dev_nSpks, sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(spkTimes, dev_spkTimes, MAX_SPKS * sizeof(float), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(spkNeuronIds, dev_spkNeuronIds, MAX_SPKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(vm, dev_vm, nSteps * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(host_isynap, dev_isynap, nSteps * N_NEURONS * sizeof(float), hipMemcpyDeviceToHost));
  /* ================= RECORD COMPUTE TIME ====================================================*/
  hipEventRecord(stop0, 0);
  hipEventSynchronize(stop0);
  printf(" Done ! \n");
  if((devErr = hipEventElapsedTime(&elapsedTime, start0, stop0)) == hipSuccess) {
    printf("\n elapsed time = %fms \n", elapsedTime);
  }
  cudaCheck(hipEventDestroy(start0));
  cudaCheck(hipEventDestroy(stop0));
  printf("\n nSpks = %d\n", *nSpks);
  printf(" MAX SPKS stored on GPU = %d \n", MAX_SPKS); 
  printf("\n Simulation completed ! \n");
  /* ================= SAVE TO DISK =============================================================*/
  printf(" saving results to disk ..."); 
  fflush(stdout);
  fp = fopen("vm", "w");
  for(i = 0; i < nSteps; ++i) {
    for(k = 0; k < N_NEURONS; ++k) {
      fprintf(fp, "%f %f ", vm[k + i *  N_NEURONS], host_isynap[k + i * N_NEURONS]);
    }
    fprintf(fp, "\n");
  }
  fpConMat = fopen("conMat.csv", "w");
  for(i = 0; i < N_NEURONS; ++i) {
    for(k = 0; k < N_NEURONS; ++k) {
      fprintf(fpConMat, "%d ", conVec[i + N_NEURONS *k]);
    }
    fprintf(fpConMat, "\n");
  }
  fpSpkTimes = fopen("spkTimes.csv", "w");
  for(i = 0; i < *nSpks; ++i) {
    fprintf(fpSpkTimes, "%f %d\n", spkTimes[i], spkNeuronIds[i]);
  }
  printf("Done!\n");  
  /*================== CLEANUP ===================================================================*/
  fclose(fp);
  cudaCheck(hipHostFree(vm));
  cudaCheck(hipHostFree(host_isynap));
  cudaCheck(hipHostFree(spkTimes));
  cudaCheck(hipHostFree(spkNeuronIds));
  cudaCheck(hipHostFree(nSpks));
  /*  cudaCheck(hipHostFree(vstart));*/
  cudaCheck(hipFree(dev_vm));
  cudaCheck(hipFree(dev_isynap));
  cudaCheck(hipFree(dev_spkNeuronIds));
  cudaCheck(hipFree(dev_spkTimes));
  cudaCheck(hipFree(dev_conVec));
  /*  cudaCheck(hipFree(dev_vstart));*/
  cudaCheck(hipFree(dev_nSpks));
  cudaCheck(hipFree(devStates));
  return EXIT_SUCCESS;
}

