#ifndef _AUX_
#define _AUX_
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "globalVars.h"
#include "devFunctionProtos.h"
#include "cudaRandFuncs.cu"
/* #include "rkdumb.cu" // NVIDIA provides no linker so have to include SOURCE FILES to keep files of managble size */
#include "isynap.cu"
#include "rkdumbPretty.cu"
#include "GenSparseMat.cu"
#include "bgCurrent.cu"
#include "IFF.cu"
#endif
