#include "hip/hip_runtime.h"
#ifndef _CUDARANDFUNCS_
#define _CUDARANDFUNCS_
#include <hip/hip_runtime.h>
//#include "hiprand/hiprand_kernel.h"
#include "mycurand.h"
#include "devFunctionProtos.h"
#include "devHostConstants.h"

__global__ void setup_kernel(hiprandState *state, unsigned long long seed ) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    if(id < N_NEURONS) {
      hiprand_init(seed * (id + 7), id, 0, &state[id]);
    }
}

__device__ float randkernel(hiprandState *state) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  float randNumber;
  hiprandState localState = state[id]; // state in global memory 
  randNumber = hiprand_uniform(&localState);
  state[id] = localState;
  return randNumber;
}


__global__ void kernelGenConMat(hiprandState *state, int *dev_conVec){
  /* indexing of matrix row + clm x N_NEURONS*/
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  float k, n;
  
  if(id < N_NEURONS) {
    k = (float)K;
    /* E --> EI */
    if(id < NE & NE > 0) {
      n = (float)NE;
      for(i = 0; i < N_NEURONS; ++i) {
        if(i < NE) {  /* E --> E */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 1;
          } 
        }
        if(i > NE) { /* E --> I */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 1;
          } 
        }
      }
    }

    /* I --> EI */
    if(id > NE & NI > 0) {
      n = (float)NI;
      for(i = 0; i < N_NEURONS; ++i) {
        if(i < NE) {  /* I --> E */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 4;
          } 
        }
        if(i > NE) { /* I --> I */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 1;
          } 
        }
      }
    }
  }
}
#endif
