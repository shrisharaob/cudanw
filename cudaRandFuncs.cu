#include "hip/hip_runtime.h"
#ifndef _CUDARANDFUNCS_
#define _CUDARANDFUNCS_
#include <hip/hip_runtime.h>
#include "globalVars.h"
/*#include "hiprand/hiprand_kernel.h"*/
#include "mycurand.h"
#include "devFunctionProtos.h"
#include "devHostConstants.h"

__global__ void setup_kernel(hiprandState *state, unsigned long long seed ) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    if(id < N_NEURONS) {
      hiprand_init(seed * (id + 7), id, 0, &state[id]);
    }
}

__global__ void setup_pois_kernel(hiprandState *state, unsigned long long seed ) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    if(id < NFF) { // <==== the only difference as compared to the previos set uo func
      hiprand_init(seed * (id + 7), id, 0, &state[id]);
    }
}

__device__ double randkernel(hiprandState *state) {
  /*RETURNS ONE SAMPLE FROM UNIFORM DISTRIBUTION*/
  unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
  float randNumber;
  hiprandState localState = state[id]; /* state in global memory */
  randNumber = hiprand_uniform(&localState);
  state[id] = localState;
  return (double)randNumber;
}

__device__ double normRndKernel(hiprandState *state) {
  /*RETURNS ONE SAMPLE FROM UNIFORM DISTRIBUTION*/
  unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
  float randNumber;
  hiprandState localState = state[id]; /* state in global memory */
  randNumber = hiprand_normal(&localState);
  state[id] = localState;
  return (double)randNumber;
}
/* same as setup_kernel, but avoids passing device pointers from host */
__global__ void setupBGCurGenerator(unsigned long long seed) {
  unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id < N_Neurons) {
    hiprand_init(seed * (id + 23), id, 0, &bgCurNoiseGenState[id]);
  }
}

__global__ void setupIFFRndGenerator(unsigned long long seed) {
  unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id < N_Neurons) {
    hiprand_init(seed * (id + 7), id, 0, &iffNormRandState[id]);
  }
}

__global__ void kernelGenConMat(hiprandState *state, int *dev_conVec){
  /* indexing of matrix row + clm x N_NEURONS*/
  unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  float k, n;
  
  if(id < N_NEURONS) {
    k = (float)K;
    /* E --> EI */
    if(id < NE & NE > 0) {
      n = (float)NE;
      for(i = 0; i < N_NEURONS; ++i) {
        if(i < NE) {  /* E --> E */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 1;
          }
        }
        if(i >= NE) { /* E --> I */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 1;
          } 
        }
      }
    }
    /* I --> EI */
    if(id >= NE & NI > 0) {
      n = (float)NI;
      for(i = 0; i < N_NEURONS; ++i) {
        if(i < NE) {  /* I --> E */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 1;
          } 
        }
        if(i >= NE) { /* I --> I */
          if(k/n >= randkernel(state)) { /* neuron[id] receives input from i ? */
            dev_conVec[id + i * N_NEURONS] = 1;
          } 
        }
      }
    }
  }
}
#endif
