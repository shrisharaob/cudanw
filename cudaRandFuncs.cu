#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include "hiprand/hiprand_kernel.h"
#include "mycurand.h"
#include "devFunctionProtos.h"
#include "devHostConstants.h"

__global__ void setup_kernel(hiprandState *state, unsigned long long seed ) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    if(id < N_NEURONS) {
      hiprand_init(seed * (id + 7), id, 0, &state[id]);
    }
}

__device__ float randkernel(hiprandState *state) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  float randNumber;
  hiprandState localState = state[id]; // state in global memory 
  randNumber = hiprand_uniform(&localState);
  state[id] = localState;
  return randNumber;
}


__global__ void kernelGenConMat(hiprandState *state, int *dev_conVec){
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  float k, n;
  if(id < N_NEURONS) {
    k = (float)K;
    n = (float)N_NEURONS;
    for(i = 0; i < N_NEURONS; ++i) {
      if(k/n >= randkernel(state)) { // neuron[id] receives input from i ?
        dev_conVec[id + i * N_NEURONS] = 1;
      }
    }
  }
}
