#include "hip/hip_runtime.h"
#ifndef _ISYNAP_
#define _ISYNAP_
#include  <hip/hip_runtime.h>
#include "globalVars.h"
#include "devHostConstants.h"
#include "devFunctionProtos.h"
#define MAX_SPKS_PER_T_STEP 1000

__device__ float isynap(float vm, int *dev_conVec) {
  int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int i, spkNeuronId[MAX_SPKS_PER_T_STEP], localNSpks = 0;
  float totIsynap = 0, gE, gI, tempCurE = 0, tempCurI = 0;
  /* compute squares of entries in data array */
  /*!!!!! neurons ids start from ZERO  !!!!!! */
  if(mNeuron < N_NEURONS) {
    gE = dev_gE[mNeuron];
    gI = dev_gI[mNeuron];
    gE *= EXP_SUM;
    gI *= EXP_SUM;
    for(i = 0; i < N_NEURONS; ++i) {
      if(dev_IF_SPK[i]) { /* too many global reads */
        spkNeuronId[localNSpks] = i; 
        localNSpks += 1; /* nspks in prev step*/
      }
    }
    if(localNSpks > 0){
      for(i = 0; i < localNSpks; ++i) { 
        if(spkNeuronId[i] < NE) {
          gE += dev_conVec[spkNeuronId[i] + N_NEURONS * mNeuron];
        }
        else {
          gI += dev_conVec[spkNeuronId[i] + N_NEURONS * mNeuron]; /*optimize !!!! gEI_I*/
        }
      }
    }
    dev_gE[mNeuron] = gE;
    dev_gI[mNeuron] = gI;
    if(mNeuron < NE) {
      tempCurE = -1 *  gE * (1/sqrt(K)) * INV_TAU_SYNAP * G_EE
                          * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
      tempCurI = -1 * gI * (1/sqrt(K)) * INV_TAU_SYNAP * G_EI
                          * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
    }
    if(mNeuron >= NE){
      tempCurE = -1 * gE * (1/sqrt(K)) * INV_TAU_SYNAP * G_IE * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
      tempCurI = -1 * gI * (1/sqrt(K)) * INV_TAU_SYNAP * G_II * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
    }
    totIsynap = tempCurE + tempCurI; 
  }
  return totIsynap;
}

__device__ float SparseIsynap(double vm, int *dev_nPostNeurons, int *dev_sparseConVec, int *dev_sparseIdx, int IF_SPK) {
  int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int kNeuron, localCurConter;
  float totIsynap = 0, gE, gI, tempCurE = 0, tempCurI = 0;
  if(mNeuron < N_NEURONS) {
    dev_gE[mNeuron] *= EXP_SUM;
    dev_gI[mNeuron] *= EXP_SUM;
     if(IF_SPK) {  
      for(kNeuron = 0; kNeuron < dev_nPostNeurons[mNeuron]; ++kNeuron) { 
        if(mNeuron < NE) {       
          atomicAdd(&dev_gE[dev_sparseConVec[dev_sparseIdx[mNeuron] + kNeuron]], 1.0f); /*atomic float add WORKS ONLY ON CC >= 2.0 */
       }
        else
          atomicAdd(&dev_gI[dev_sparseConVec[dev_sparseIdx[mNeuron] + kNeuron]], 1.0f);
      }
     }
      __syncthreads();
     gE = dev_gE[mNeuron];
     gI = dev_gI[mNeuron];
     if(mNeuron < NE) {
       tempCurE = -1 * gE * (1/sqrt(K)) * INV_TAU_SYNAP * G_EE * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
       tempCurI = -1 * gI * (1/sqrt(K)) * INV_TAU_SYNAP * G_EI * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
     }
     if(mNeuron >= NE){
       tempCurE = -1 * gE * (1/sqrt(K)) * INV_TAU_SYNAP * G_IE * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
       tempCurI = -1 * gI * (1/sqrt(K)) * INV_TAU_SYNAP * G_II * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
     }
     totIsynap = tempCurE + tempCurI; 
     if(mNeuron == 16003) {
       localCurConter = curConter;
       if(curConter < 5 * 4000) {
	 glbCurE[localCurConter] = tempCurE;
	 glbCurI[localCurConter] = tempCurI;
	 curConter += 1;
       }
     }
  }
  return totIsynap;
}
#endif
