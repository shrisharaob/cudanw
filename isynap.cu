#include "hip/hip_runtime.h"
#ifndef _ISYNAP_
#define _ISYNAP_
#include  <hip/hip_runtime.h>
#include "globalVars.h"
#include "devHostConstants.h"
#include "devFunctionProtos.h"
#define MAX_SPKS_PER_T_STEP 1000

__device__ double isynap(double vm, int *dev_conVec) {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int i, spkNeuronId[MAX_SPKS_PER_T_STEP], localNSpks = 0;
  double totIsynap = 0, gE, gI, tempCurE = 0, tempCurI = 0;
  /* compute squares of entries in data array */
  /*!!!!! neurons ids start from ZERO  !!!!!! */
  if(mNeuron < N_NEURONS) {
    gE = dev_gE[mNeuron];
    gI = dev_gI[mNeuron];
    gE *= EXP_SUM;
    gI *= EXP_SUM;
    for(i = 0; i < N_NEURONS; ++i) {
      if(dev_IF_SPK[i]) { /* too many global reads */
        spkNeuronId[localNSpks] = i; 
        localNSpks += 1; /* nspks in prev step*/
      }
    }
    if(localNSpks > 0){
      for(i = 0; i < localNSpks; ++i) { 
        if(spkNeuronId[i] < NE) {
          gE += dev_conVec[spkNeuronId[i] + N_NEURONS * mNeuron];
        }
        else {
          gI += dev_conVec[spkNeuronId[i] + N_NEURONS * mNeuron]; /*optimize !!!! gEI_I*/
        }
      }
    }
    dev_gE[mNeuron] = gE;
    dev_gI[mNeuron] = gI;
    if(mNeuron < NE) {
      tempCurE = -1 *  gE * (1/sqrt(K)) * INV_TAU_SYNAP * G_EE
                          * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
      tempCurI = -1 * gI * (1/sqrt(K)) * INV_TAU_SYNAP * G_EI
                          * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
    }
    if(mNeuron >= NE){
      tempCurE = -1 * gE * (1/sqrt(K)) * INV_TAU_SYNAP * G_IE * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
      tempCurI = -1 * gI * (1/sqrt(K)) * INV_TAU_SYNAP * G_II * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
    }
    totIsynap = tempCurE + tempCurI; 
  }
  return totIsynap;
}

__global__ void expDecay() {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  if(mNeuron < N_NEURONS) {
    dev_gE[mNeuron] *= EXP_SUM;
    dev_gI[mNeuron] *= EXP_SUM;
  }
}

__global__ void expDecay(int *dev_histCountE, int *dev_histCountI) {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;
  while(mNeuron < N_NEURONS) {
    dev_gE[mNeuron] *= EXP_SUM;
    dev_gI[mNeuron] *= EXP_SUM;
    /*    if(mNeuron == 0) {
      for(int i = 0; i < N_NEURONS; ++i) {*/
    dev_histCountE[mNeuron] = 0;
    dev_histCountI[mNeuron] = 0;
        /*      }
    }*/
    mNeuron += stride;
  }
}


__global__ void computeConductanceHist(int *dev_histCountE, int *dev_histCountI) {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;
  while(mNeuron < N_NEURONS) {
      dev_gE[mNeuron] += (double)dev_histCountE[mNeuron];
      dev_gI[mNeuron] += (double)dev_histCountI[mNeuron];
      mNeuron += stride;
  }
}     


__global__ void computeConductanceHistFF(int *dev_histCountFF) {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;
  while(mNeuron < N_NEURONS) {
    gFF[mNeuron] += (double)dev_histCountFF[mNeuron];
    mNeuron += stride;
  }
}     



__global__ void computeConductance() {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int kNeuron;
  if(mNeuron < N_NEURONS) {
     if(dev_IF_SPK[mNeuron]) {  
      for(kNeuron = 0; kNeuron < dev_nPostNeurons[mNeuron]; ++kNeuron) { 
        if(mNeuron < NE) {       
          atomicAdd(&dev_gE[dev_sparseConVec[dev_sparseIdx[mNeuron] + kNeuron]], (double)1.0); /*atomic double add WORKS ONLY ON CC >= 2.0 */
       }
        else
          atomicAdd(&dev_gI[dev_sparseConVec[dev_sparseIdx[mNeuron] + kNeuron]], (double)1.0);
      }
     } 
  }
}

__global__ void computeG_Optimal() {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int kNeuron, localSpkId;
  if(mNeuron < N_NEURONS) {
    if(dev_IF_SPK[mNeuron]) {  
      localSpkId = dev_prevStepSpkIdx[mNeuron];
      for(kNeuron = 0; kNeuron < dev_nPostNeurons[mNeuron]; ++kNeuron) { 
        if(mNeuron < NE) {       
	  dev_ESpkCountMat[dev_sparseConVec[dev_sparseIdx[mNeuron] + kNeuron] + N_NEURONS + localSpkId] += 1;
	}
        else{
	  dev_ISpkCountMat[dev_sparseConVec[dev_sparseIdx[mNeuron] + kNeuron] + N_NEURONS + localSpkId] += 1;
	}
      }
    }
  }
}


__global__ void spkSum(int nSpksInPrevStep) {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  int i, gE, gI; 
  int stride = gridDim.x * blockDim.x;
  while(mNeuron < N_NEURONS) {
    gE = 0;
    gI = 0;
    for(i = 0; i < nSpksInPrevStep ; ++i){
      gE += dev_ESpkCountMat[mNeuron + i * N_NEURONS];
      gI += dev_ISpkCountMat[mNeuron + i * N_NEURONS];
    }	
    dev_gE[mNeuron] += (double)gE;
    dev_gI[mNeuron] += (double)gI;
    mNeuron += stride;
  }
}

__global__ void computeIsynap(double t) {
  unsigned int mNeuron = threadIdx.x + blockDim.x * blockIdx.x;
  double vm, tempCurE = 0, tempCurI = 0;
  // int localCurConter;
  if(mNeuron < N_NEURONS) {
    vm = dev_v[mNeuron];
    if(mNeuron < NE) {
      tempCurE = -1 * dev_gE[mNeuron] * (1/sqrt(K)) * INV_TAU_SYNAP * G_EE * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
      tempCurI = -1 * dev_gI[mNeuron] * (1/sqrt(K)) * INV_TAU_SYNAP * G_EI * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
    }
    if(mNeuron >= NE){
      tempCurE = -1 * dev_gE[mNeuron] * (1/sqrt(K)) * INV_TAU_SYNAP * G_IE * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));
      tempCurI = -1 * dev_gI[mNeuron] * (1/sqrt(K)) * INV_TAU_SYNAP * G_II * (RHO * (vm - V_I) + (1 - RHO) * (E_L - V_I));
    }
    dev_isynap[mNeuron] = tempCurE + tempCurI; 
    // if(mNeuron == SAVE_CURRENT_FOR_NEURON) {
    //   localCurConter = curConter;
    //   if(localCurConter < N_CURRENT_STEPS_TO_STORE) {
    //     //glbCurE[localCurConter]   = tempCurE;
    //     //glbCurI[localCurConter] = tempCurI;
    //     dev_iff[localCurConter] = dev_iffCurrent[mNeuron];
    //     curConter += 1;
    //   }
    // }
    	/* bg current */
	/*	ibg = bgCur(vmOld); /* make sure AuxRffTotal<<<  >>> is run begore calling bgCur */
	/* FF input current*/
    /*
    RffTotal(t);
    Gff(t);
    */
    /* FF current computed from the spikes of the layer 4 Poission generators  */
    
    
    dev_iffCurrent[mNeuron] = IFF_orimap(vm);
  }
}
#endif
