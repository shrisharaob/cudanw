#include "hip/hip_runtime.h"
__device__ double bgCur(double vm) {
  unsigned int mNeuron = threadIdx.x + blockIdx.x * blockDim.x;
  double D = 1, iBg = 0;
  double gE, gI, gNoise;
  if(mNeuron < N_NEURONS) {
    if(mNeuron < NE) {
      gNoise = gaussNoiseE[mNeuron];
      gNoise = gNoise * (1 - DT * INV_TAU_SYNAP) + SQRT_DT  * INV_TAU_SYNAP * normRndKernel(bgCurNoiseGenState);
      /*gNoise = 0;*/
      gE = G_EB * K * (RB_E + sqrt(RB_E / K) * gNoise);
      /*gE = G_EB * K * RB_E;*/
      /*      iBg = -1 * gE * (RHO * (vm - V_E) + (1 - RHO) * (E_L - V_E));*/
      iBg = -1 * gE * (RHO * vm + (1 - RHO) * E_L);
      gaussNoiseE[mNeuron] = gNoise;
    }
    if(mNeuron >= NE) {
      gNoise = gaussNoiseI[mNeuron - NE];
      gNoise = gNoise * (1 - DT * INV_TAU_SYNAP) +  SQRT_DT  * INV_TAU_SYNAP * normRndKernel(bgCurNoiseGenState);
      /*      gNoise = 0;*/
      gI = G_IB * K * (RB_I + sqrt(RB_I / K) * gNoise);
      /*      gI = G_IB * K * RB_I;*/
      iBg = -1 * gI * (RHO * vm + (1 - RHO) * E_L);
      gaussNoiseI[mNeuron - NE] = gNoise;
    }
    /*    if(mNeuron == SAVE_CURRENT_FOR_NEURON) {
      dev_bgCur[curConter - 1] = gNoise;
      }*/
  }
  D +=1;
  return iBg;
}
